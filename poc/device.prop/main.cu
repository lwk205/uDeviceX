
#include <hip/hip_runtime.h>
#include <stdio.h>

void GetDeviceProperties(struct hipDeviceProp_t *prop) {
    hipError_t e;
    int device;
    device = 0;
    e = hipGetDeviceProperties (prop, device);
    if (e != hipSuccess) {
        fprintf(stderr, "GetDeviceProperties failed\n");
        exit(2);
    }
}

int main() {
    hipDeviceProp_t p;
    GetDeviceProperties(&p);
    printf("totalGlobalMem = %ld\n", p.totalGlobalMem);
}
