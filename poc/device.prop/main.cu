
#include <hip/hip_runtime.h>
#include <stdio.h>

void GetDeviceProperties(struct hipDeviceProp_t *prop) {
    hipError_t e;
    int device;
    device = 0;
    e = hipGetDeviceProperties (prop, device);
    if (e != hipSuccess) {
        fprintf(stderr, "GetDeviceProperties failed\n");
        exit(2);
    }
}

int main() {
    hipDeviceProp_t p;
    GetDeviceProperties(&p);
    printf("totalGlobalMem: % 09ld\n", p.totalGlobalMem);
    printf("maxTexture1D  : % 09d\n", p.maxTexture1D);
}
