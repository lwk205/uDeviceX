/*
  hello world : add two vectors

 */


#include <hip/hip_runtime.h>
#include <stdio.h>

/* ceiling `m' to `n' (returns the smallest `A' such n*A is not less
   than `m') */
#define ceiln(m, n) (   ((m) + (n) - 1)/(n)   )

/* a common kernel execution configuration */
#define k_cnf(n) ceiln((n), 128), 128

#define n 32 /* number of elements */
char *A, *C;
int  *B;

__global__ void f (char *A, int *B, char *C) {
    int i, cnt;
    for (cnt = i = 0; i < n / 2; i++)
        C[cnt++] = B[i];

    for (i = 0; i < n / 2; i++)
        C[cnt++] = ((char*)B)[i];
}

void ini() {
    hipMalloc(&A, n*sizeof(A[0]));
    hipMalloc(&B, n*sizeof(B[0]));
    hipMalloc(&C, n*sizeof(C[0]));    
}

int main() {
  ini();
  f<<<k_cnf(n)>>>(A, B, C);
}
