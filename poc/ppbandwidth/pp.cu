#include "hip/hip_runtime.h"
#include <stdio.h>

#include "u.h" /* utils */

#define ceiln(m, n) (   ((m) + (n) - 1)/(n)   )
#define k_cnf(n) ceiln((n), 128), 128

#define dt (1e-1)

struct Particle {
    float r[3], v[3];
};

struct Particle4 {
    float4 r, v;
};

#include "vanilla.h"
#include "float4.h"
#include "P4.h"

void print_bw(const char *fun, float t, size_t nbytes, int neval, float rw) {
    double tav = t / neval;
    double bw = (nbytes * rw / tav) * 1e-6;
    printf("%20s : t = %6e [ms], %6e [GB/s]\n", fun, tav, bw);
}

#define ESC(...) __VA_ARGS__
#define measure(F, C, A, nbytes, rw) do {                               \
        CC(hipEventRecord(start));                                     \
        for (int i = 0; i < ntrials; ++i) F <<<ESC C>>> A;              \
        CC(hipEventRecord(stop));                                      \
        CC(hipEventSynchronize(stop));                                 \
        CC(hipEventElapsedTime(&t, start, stop));                      \
        print_bw(#F, t, nbytes, ntrials, rw);                           \
    } while (0)

int main() {
    int n = 100000, ntrials = 10000;

    Particle *pp;
    Particle4 *pp4;

    hipEvent_t start, stop;
    float t;

    CC(hipSetDevice(2));
    
    CC(hipMalloc(&pp,  n*sizeof(Particle)));
    CC(hipMalloc(&pp4, n*sizeof(Particle4)));

    CC(hipEventCreate(&start));
    CC(hipEventCreate(&stop));

    float rwini = 2.0/3.0, rwupd = 1.5;
    
    measure(iniP, (k_cnf(n)), (n, pp), n*sizeof(float), rwini);
    measure(updP, (k_cnf(n)), (n, pp), n*sizeof(float), rwupd);

    measure(inif, (k_cnf(n)), (n, (float*)pp), n*sizeof(Particle), rwini);
    measure(updf, (k_cnf(n)), (n, (float*)pp), n*sizeof(Particle), rwupd);

    measure(inif4, (k_cnf(n)), (n, (float4*) pp4), 2*n*sizeof(float4), rwini);
    measure(updf4, (k_cnf(n)), (n, (float4*) pp4), 2*n*sizeof(float4), rwupd);

    measure(inif4_2tpp, (k_cnf(2*n)), (n, (float4*) pp4), 2*n*sizeof(float4), rwini);
    measure(updf4_2tpp, (k_cnf(2*n)), (n, (float4*) pp4), 2*n*sizeof(float4), rwupd);

    measure(iniP4, (k_cnf(n)), (n, pp4), n*sizeof(Particle4), rwini);
    measure(updP4, (k_cnf(n)), (n, pp4), n*sizeof(Particle4), rwupd);

    CC(hipFree(pp));
    CC(hipFree(pp4));
}
