#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 10
int *a;

__global__ void uninit(int *a) {
    print
}


void run_uninit() {
    uninit<<<1,1>>>(a);
    hipGetErrorString(hipGetLastError());
    printf("Sync: %s\n", hipGetErrorString(hipDeviceSynchronize()));
}

int main() {
    hipMalloc(&a, N*sizeof(a[0]));

    run_uninit();

    hipDeviceReset();
    hipFree(a);
    return 0;
}
