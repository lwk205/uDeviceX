#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 1
int *a;

__global__ void uninit(int *a) {
    printf("a[0]: %d\n", a[0]);
}


void run_uninit() {
    uninit<<<1,1>>>(a);
    hipGetErrorString(hipGetLastError());
    printf("Sync: %s\n", hipGetErrorString(hipDeviceSynchronize()));
}

int main() {
    hipMalloc(&a, N*sizeof(a[0]));

    run_uninit();

    hipDeviceReset();
    hipFree(a);
    return 0;
}
