#include "hip/hip_runtime.h"
#include "mesh.h"
#include "collision.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define CC(ans)                                             \
    do { cudaAssert((ans), __FILE__, __LINE__); } while (0)
inline void cudaAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        abort();
    }
}

#define DEVICE

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost

// #define VANILLA
// #define SHARED
// #define TEXTURE
#define TEXTURE_SHARED

#include "tex.h"

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        fprintf(stderr, "Usage: %s <N> <file.ply>\n", argv[0]);
        exit(1);
    }

    const int N = atoi(argv[1]);
    
    srand48(123456);
    
    std::vector<int> tt;
    std::vector<float> vv;

    mesh::read_ply(argv[2], tt, vv);

    const int nv = vv.size() / 3;
    const int nt = tt.size() / 3;
    
    // generate points
    
    float xlo, xhi, ylo, yhi, zlo, zhi;

    xlo = xhi = vv[0];
    ylo = yhi = vv[1];
    zlo = zhi = vv[2];
    
    for (int i = 0; i < nv; ++i)
    {
        const float x = vv[3*i + 0], y = vv[3*i + 1], z = vv[3*i + 2];

#define highest(a, b) do {a = a < b ? b : a; } while(0)
#define  lowest(a, b) do {a = a < b ? a : b; } while(0)

        lowest(xlo, x); highest(xhi, x);
        lowest(ylo, y); highest(yhi, y);
        lowest(zlo, z); highest(zhi, z);
    }

    printf("Extents: %f %f, %f %f, %f %f\n", xlo, xhi, ylo, yhi, zlo, zhi);

    float *rr  = new float[3*N];
    int *inout = new int[N];

    for (int i = 0; i < N; ++i)
    {
        rr[i*3 + 0] = xlo + drand48() * (xhi - xlo);
        rr[i*3 + 1] = ylo + drand48() * (yhi - ylo);
        rr[i*3 + 2] = zlo + drand48() * (zhi - zlo);
    }
    
    // compute inout

#ifdef DEVICE

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float *d_vv = NULL, *d_rr = NULL; int *d_tt = NULL, *d_inout = NULL;
    CC(hipMalloc(&d_vv, 3 * nv * sizeof(float)));
    CC(hipMalloc(&d_tt, 3 * nt * sizeof(int)));
    CC(hipMalloc(&d_rr, 3 * N  * sizeof(float)));
    CC(hipMalloc(&d_inout, N   * sizeof(int)));
    
    CC(hipMemcpy(d_vv, vv.data(), 3 * nv * sizeof(float), H2D));
    CC(hipMemcpy(d_tt, tt.data(), 3 * nt * sizeof(int),   H2D));
    CC(hipMemcpy(d_rr, rr, 3 * N * sizeof(float), H2D));

#if defined( TEXTURE ) || defined( TEXTURE_SHARED )
    float4 *vvzip; int4 *ttzip;
    CC(hipMalloc(&vvzip, nv * sizeof(float4)));
    CC(hipMalloc(&ttzip, nt * sizeof(int4)));

    tex::zip4(d_tt, nt, /**/ ttzip);
    tex::zip4(d_vv, nv, /**/ vvzip);

    hipTextureObject_t ttto, vvto;

    tex::maketexzip(ttzip, nt, /**/ &ttto);
    tex::maketexzip(vvzip, nv, /**/ &vvto);
    
#endif
    
    hipEventRecord(start);

#if defined( VANILLA )
    collision::in_mesh_dev(d_rr, N, d_vv, nv, d_tt, nt, /**/ d_inout);
#elif defined( SHARED )
    collision::in_mesh_dev_shared(d_rr, N, d_vv, nv, d_tt, nt, /**/ d_inout);
#elif defined( TEXTURE )
    collision::in_mesh_dev_tex(d_rr, N, vvto, nv, ttto, nt, /**/ d_inout);
#elif defined( TEXTURE_SHARED )
    collision::in_mesh_dev_tex_shared(d_rr, N, vvto, nv, ttto, nt, /**/ d_inout);
#endif

    hipEventRecord(stop);
    
    CC(hipMemcpy(inout, d_inout, N * sizeof(int), D2H));
    
    CC(hipFree(d_vv)); CC(hipFree(d_tt));
    CC(hipFree(d_rr)); CC(hipFree(d_inout));

#if defined( TEXTURE ) || defined( TEXTURE_SHARED )
    CC(hipFree(vvzip)); CC(hipFree(ttzip));
#endif
    
    hipEventSynchronize(stop);
    float tms = 0;
    hipEventElapsedTime(&tms, start, stop);
    fprintf(stderr, "Took %f ms for %d particles\n", tms, N);
#else
    collision::in_mesh(rr, N, vv.data(), tt.data(), nt, /**/ inout);
#endif
    
    // dump
    
    FILE *fin = fopen("parts_in.3D", "w");
    FILE *fout = fopen("parts_out.3D", "w");

    fprintf(fin, "x y z inout\n");
    fprintf(fout, "x y z inout\n");
    
    for (int i = 0; i < N; ++i)
    fprintf(inout[i] == -1 ? fout : fin, "%.6e %.6e %.6e %e\n", rr[3*i + 0], rr[3*i + 1], rr[3*i + 2], inout[i] == -1 ? 1.f : 0.f);

    delete[] inout;
    delete[] rr;
    
    fclose(fin);
    fclose(fout);
}

/*

# nTEST: collision.t0
# make clean && make -j
# ./inmesh 10000 data/cow.ply
# cat parts_in.3D | sed -n '2,10000p' > parts.out.3D

# nTEST: collision.t1
# make clean && make -j
# ./inmesh 10000 data/sphere.ply
# cat parts_in.3D | sed -n '2,10000p' > parts.out.3D

*/
