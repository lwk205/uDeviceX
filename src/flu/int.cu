#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <conf.h>
#include "inc/conf.h"
#include "d.h"
#include "common.h"
#include "msg.h"
#include "cc.h"
#include "inc/type.h"
#include "inc/dev.h"
#include "clist/int.h"
#include "rnd/imp.h"

#include "flu/int.h"
#include "flu/imp.h"

namespace flu {

void alloc_quants(Quants *q) {
    q->n = 0;
    Dalloc0(&q->pp, MAX_PART_NUM);
    Dalloc0(&q->pp0, MAX_PART_NUM);
    q->cells = new clist::Clist(XS, YS, ZS);
    q->pp_hst = new Particle[MAX_PART_NUM];
}

void free_quants(Quants *q) {
    CC(hipFree(q->pp)); CC(hipFree(q->pp0));
    delete q->cells;
    delete[] q->pp_hst;
}

void alloc_quantsI(QuantsI *q) {
    Dalloc(&q->ii, MAX_PART_NUM);
    Dalloc(&q->ii0, MAX_PART_NUM);
    q->ii_hst = new int[MAX_PART_NUM];
}

void free_quantsI(QuantsI *q) {
    CC(hipFree(q->ii)); CC(hipFree(q->ii0));
    delete[] q->ii_hst;
}

void alloc_ticketZ(/**/ TicketZ *t) {
    Dalloc0(&t->zip0, MAX_PART_NUM);
    Dalloc0(&t->zip1, MAX_PART_NUM);
}

void free_ticketZ(/**/ TicketZ *t) {
    float4  *zip0 = t->zip0;
    ushort4 *zip1 = t->zip1;
    hipFree(zip0);
    hipFree(zip1);
}

void get_ticketZ(Quants q, /**/ TicketZ *t) {
    if (q.n == 0) return;
    float4  *zip0 = t->zip0;
    ushort4 *zip1 = t->zip1;
    sub::zip(q.pp, q.n, /**/ zip0, zip1);
}

void get_ticketRND(/**/ TicketRND *t) {
    t->rnd = new rnd::KISS(0, 0, 0, 0);
}

void free_ticketRND(/**/ TicketRND *t) {
    delete t->rnd;
}

void gen_quants(Quants *q) {
    q->n = sub::gen(q->pp, q->pp_hst);
}

void gen_ids(const int n, QuantsI *q) {
    sub::ii_gen(n, q->ii, q->ii_hst);
}

void gen_tags0(const int n, QuantsI *q) {
    sub::tags0_gen(n, q->ii, q->ii_hst);
}

void strt_quants(const int id, Quants *q) {
    q->n = sub::strt(id, /**/ q->pp, /* w */ q->pp_hst);
}

void strt_ii(const char *subext, const int id, QuantsI *q) {
    sub::strt_ii(subext, id, /**/ q->ii, /* w */ q->ii_hst);
}

void strt_dump(const int id, const Quants q) {
    sub::strt_dump(id, q.n, q.pp, /* w */ q.pp_hst);
}

void strt_dump_ii(const char *subext, const int id, const QuantsI q, const int n) {
    sub::strt_dump_ii(subext, id, n, q.ii, /* w */ q.ii_hst);
}

} // flu
