#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

#include <mpi.h>
#include "l/m.h"
#include "m.h"
#include "inc/type.h"
#include "common.h"
#include "common.cuda.h"
#include "common.mpi.h"
#include <conf.h>
#include "restart.h"

#include "flu/imp.h"
#include "flu/dev.h"

namespace flu {
namespace sub {

static int gen0(Particle *pp) { /* generate particle positions and velocities */
    enum {X, Y, Z};
    assert(XS * YS * ZS * numberdensity < MAX_PART_NUM);
  
    srand48(123456);
    int iz, iy, ix, l, nd = numberdensity;
    int n = 0; /* particle index */
    float x, y, z, dr = 0.99;
    for (iz = 0; iz < ZS; iz++)
    for (iy = 0; iy < YS; iy++)
    for (ix = 0; ix < XS; ix++) {
        /* edge of a cell */
        int xlo = -0.5*XS + ix, ylo = -0.5*YS + iy, zlo = -0.5*ZS + iz;
        for (l = 0; l < nd; l++) {
            Particle p = Particle();
            x = xlo + dr * drand48(), y = ylo + dr * drand48(), z = zlo + dr * drand48();
            p.r[X] = x; p.r[Y] = y; p.r[Z] = z;

            p.v[X] = 0; p.v[Y] = 0; p.v[Z] = 0;
          
            pp[n++] = p;
        }
    }

    MSG("ic::gen: created %06d solvent particles", n);
    return n;
}

int gen(Particle *dev, /*w*/ Particle *hst) {
    int n = gen0(hst);
    cH2D(dev, hst, n);
    return n;
}

static void ii_gen0(const long n, int *ii) {
    long i0 = 0;
    MC(l::m::Exscan(&n, &i0, 1, MPI_LONG, MPI_SUM, m::cart));
    for (long i = 0; i < n; ++i) ii[i] = i + i0;
}

void ii_gen(const int n, int *ii_dev, int *ii_hst) {
    ii_gen0(n, ii_hst);
    cH2D(ii_dev, ii_hst, n);
}

static void tags0_gen0(const long n, int *ii) {
    for (long i = 0; i < n; ++i) ii[i] = -1;
}

void tags0_gen(const int n, int *ii_dev, int *ii_hst) {
    tags0_gen0(n, ii_hst);
    cH2D(ii_dev, ii_hst, n);
}

int strt(const int id, Particle *dev, /*w*/ Particle *hst) {
    int n;
    restart::read_pp("flu", id, hst, &n);
    if (n) cH2D(dev, hst, n);
    return n;
}

int strt_ii(const char *subext, const int id, int *dev, /*w*/ int *hst) {
    int n;
    restart::read_ii("flu", subext, id, hst, &n);
    if (n) cH2D(dev, hst, n);
    return n;
}

void strt_dump(const int id, const int n, const Particle *dev, Particle *hst) {
    if (n) cD2H(hst, dev, n);
    restart::write_pp("flu", id, hst, n);
}

void strt_dump_ii(const char *subext, const int id, const int n, const int *dev, int *hst) {
    if (n) cD2H(hst, dev, n);
    restart::write_ii("flu", subext, id, hst, n);
}

void zip(const Particle *pp, const int n, /**/ float4 *zip0, ushort4 * zip1) {
    assert(sizeof(Particle) == 6 * sizeof(float)); /* :TODO: implicit dependency */
    dev::zip<<<(n + 1023) / 1024, 1024, 1024 * 6 * sizeof(float)>>>(zip0, zip1, (float*)pp, n);
}

} // sub
} // flu
