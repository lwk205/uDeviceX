#include <stdio.h>
#include "msg.h"
#include "cc/common.h"
namespace cc {
void check(hipError_t rc, const char *file, int line) {
    if (rc != hipSuccess)
        ERR("%s:%d: %s", file, line, hipGetErrorString(rc));
}
}
