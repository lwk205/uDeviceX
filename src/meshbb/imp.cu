#include "hip/hip_runtime.h"
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"
#include "inc/dev.h"
#include "utils/kl.h"
#include "utils/cc.h"
#include "inc/type.h"
#include "d/api.h"

#include "imp.h"

namespace meshbb {

/* conf */
enum {MAX_COL = 4};
#define dbg_output

#include "bbstates.h"
#include "dbg.h"
#include "dev/roots.h"
#include "dev/utils.h"
#include "dev/intersection.h"
#include "dev/main.h"

void ini(int maxpp, /**/ BBdata *d) {
    CC(d::Malloc((void**) d->ncols,   maxpp * sizeof(int)));
    CC(d::Malloc((void**) d->datacol, maxpp * MAX_COL * sizeof(float4)));
    CC(d::Malloc((void**) d->idcol,   maxpp * MAX_COL * sizeof(int)));
}

void fin(/**/ BBdata *d) {
    CC(d::Free(d->ncols));
    CC(d::Free(d->datacol));
    CC(d::Free(d->idcol));    
}

void reini(int n, /**/ BBdata d) {
    CC(d::MemsetAsync(d.ncols, 0, n * sizeof(int)));
}

void find_collisions(int nm, int nt, const int4 *tt, const Particle *i_pp, int3 L,
                     const int *starts, const int *counts, const Particle *pp, const Force *ff,
                     /**/ BBdata d) {
    dbg::ini_dev();
    
    if (!nm) return;

    KL(dev::find_collisions, (k_cnf(nm * nt)),
       (nm, nt, tt, i_pp, L, starts, counts, pp, ff, /**/ d.ncols, d.datacol, d.idcol));

    dbg::report_dev();
}

void select_collisions(int n, /**/ BBdata d) {
    KL(dev::select_collisions, (k_cnf(n)), (n, /**/ d.ncols, d.datacol, d.idcol));
}


void bounce(int n, BBdata d, const Force *ff, int nt, const int4 *tt, const Particle *i_pp, /**/ Particle *pp, Momentum *mm) {
    
    KL(dev::perform_collisions, (k_cnf(n)),
       (n, d.ncols, d.datacol, d.idcol, ff, nt, tt, i_pp, /**/ pp, mm));
}

} // meshbb
