#include "hip/hip_runtime.h"
#include <mpi.h>
#include "common.h"
#include "minmax.h"
#include <conf.h>
#include "conf.common.h"

#include "collision.h"

namespace collision
{
enum {X, Y, Z};
#define _HD_ __host__ __device__

#define dot(x, y) (x[0]*y[0] + x[1]*y[1] + x[2]*y[2])
static void project_t(const float *a, const float *b, const float *c, const float *r, /**/ float *p) {
    const float ab[3] = {b[0]-a[0], b[1]-a[1], b[2]-a[2]};
    const float ac[3] = {c[0]-a[0], c[1]-a[1], c[2]-a[2]};
    const float ar[3] = {r[0]-a[0], r[1]-a[1], r[2]-a[2]};

    float n[3] = {ab[1]*ac[2] - ab[2]*ac[1],
                  ab[2]*ac[0] - ab[0]*ac[2],
                  ab[0]*ac[1] - ab[1]*ac[0]};
    {
        const float s = 1.f / sqrt(dot(n,n));
        n[0] *= s; n[1] *= s; n[2] *= s;
    }
        
    const float arn = dot(ar, n);
    const float g[3] = {ar[0] - arn * n[0],
                        ar[1] - arn * n[1],
                        ar[2] - arn * n[2]};
    
    float u, v;
    {
        const float ga1 = dot(g, ab);
        const float ga2 = dot(g, ac);
        const float a11 = dot(ab, ab);
        const float a12 = dot(ab, ac);
        const float a22 = dot(ac, ac);

        const float fac = 1.f / (a11*a22 - a12*a12);
            
        u = (ga1 * a22 - ga2 * a12) * fac;
        v = (ga2 * a11 - ga1 * a12) * fac;
    }
        
    // project (u,v) onto unit triangle
    
    if ( (v > u - 1) && (v < u + 1) && (v > 1 - u) ) {
        const float a_ = 0.5f * (u + v - 1);
        u -= a_;
        v -= a_;
    }
    else {
        u = max(min(1.f, u), 0.f);
        v = max(min(v, 1.f-u), 0.f);
    }
    
    // compute projected point
    p[0] = a[0] + u * ab[0] + v * ac[0];
    p[1] = a[1] + u * ab[1] + v * ac[1];
    p[2] = a[2] + u * ab[2] + v * ac[2];
}

static float dist_from_triangle(const float *a, const float *b, const float *c, const float *r) {
    float p[3];
    project_t(a, b, c, r, /**/ p);
    //for (int k = 0; k < 3; ++k) p[k] = (a[k] + b[k] + c[k]) / 3.f;
    
    const float dr[3] = {p[0] - r[0], p[1] - r[1], p[2] - r[2]};
    return sqrt(dot(dr, dr));
}
#undef dot

float dist_from_mesh(const Mesh m, const float *r0) {
    float dmin = 1e5f;

    for (int it = 0; it < m.nt; ++it) {
        const int i1 = m.tt[3*it + 0];
        const int i2 = m.tt[3*it + 1];
        const int i3 = m.tt[3*it + 2];

        const float *A = m.vv + 3*i1;
        const float *B = m.vv + 3*i2;
        const float *C = m.vv + 3*i3;

        const float d = dist_from_triangle(A, B, C, r0);

        dmin = min(d, dmin);
    }
    return dmin;
}

static _HD_ bool same_side(const float *x, const float *p, const float *a, const float *b, const float *inplane) {
    const float n[3] = {a[Y] * b[Z] - a[Z] * b[Y],
                        a[Z] * b[X] - a[X] * b[Z],
                        a[X] * b[Y] - a[Y] * b[X]};

    const float ndx = n[X] * (x[X] - inplane[X]) + n[Y] * (x[Y] - inplane[Y]) + n[Z] * (x[Z] - inplane[Z]);
    const float ndp = n[X] * (p[X] - inplane[X]) + n[Y] * (p[Y] - inplane[Y]) + n[Z] * (p[Z] - inplane[Z]);
        
    return ndx * ndp > 0;
}
    
static _HD_ bool in_tetrahedron(const float *x, const float *A, const float *B, const float *C, const float *D) {
    const float AB[3] = {B[X] - A[X], B[Y] - A[Y], B[Z] - A[Z]};
    const float AC[3] = {C[X] - A[X], C[Y] - A[Y], C[Z] - A[Z]};
    const float AD[3] = {D[X] - A[X], D[Y] - A[Y], D[Z] - A[Z]};
    
    const float BC[3] = {C[X] - B[X], C[Y] - B[Y], C[Z] - B[Z]};
    const float BD[3] = {D[X] - B[X], D[Y] - B[Y], D[Z] - B[Z]};

    return
        same_side(x, A, BC, BD, D) &&
        same_side(x, B, AD, AC, D) &&
        same_side(x, C, AB, BD, D) &&
        same_side(x, D, AB, AC, A);
}
    
int inside_1p(const float *r, const float *vv, const int *tt, const int nt) {
    int c = 0;
    float origin[3] = {0, 0, 0};
#ifdef spdir
    origin[spdir] = r[spdir];
#endif
        
    for (int i = 0; i < nt; ++i) {
        const int *t = tt + 3 * i;
        if (in_tetrahedron(r, vv + 3*t[0], vv + 3*t[1], vv + 3*t[2], origin)) ++c;
    }
    return c%2;
}

void inside_hst(const Particle *pp, const int n, const Mesh m, /**/ int *inout) {
    for (int i = 0; i < n; ++i)
    inout[i] = inside_1p(pp[i].r, m.vv, m.tt, m.nt);
}

static int inside_1p(const float *r, const Particle *vv, const int *tt, const int nt) {
    int c = 0;
    float origin[3] = {0, 0, 0};
#ifdef spdir
    origin[spdir] = r[spdir];
#endif

    for (int i = 0; i < nt; ++i) {
        const int *t = tt + 3 * i;
        if (in_tetrahedron(r, vv[t[0]].r, vv[t[1]].r, vv[t[2]].r, origin)) ++c;
    }
    return c%2;
}
    
void inside_hst(const Particle *pp, const int n, const Mesh m, const Particle *i_pp, const int ns, /**/ int *tags) {
    for (int i = 0; i < n; ++i) {
        tags[i] = -1;
        for (int sid = 0; sid < ns; ++sid)
        if (inside_1p(pp[i].r, i_pp + m.nv * sid, m.tt, m.nt)) {
            tags[i] = sid;
            break;
        }
    }
}

namespace kernels
{
__global__ void init_tags(const int n, /**/ int *tags) {
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < n) tags[gid] = -1;
}

// assume ns blocks along y
__global__ void compute_tags(const Particle *pp, const int n, const Particle *vv, const int nv, const int *tt, const int nt, /**/ int *tags) {
    const int sid = blockIdx.y;
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;

    int count = 0;

    const Particle p = pp[gid];
    float origin[3] = {0, 0, 0};
#ifdef spdir
    origin[spdir] = p.r[spdir];
#endif

        
    for (int i = 0; i < nt; ++i) {
        const int t1 = sid * nv + tt[3*i + 0];
        const int t2 = sid * nv + tt[3*i + 1];
        const int t3 = sid * nv + tt[3*i + 2];

        const float a[3] = {vv[t1].r[0], vv[t1].r[1], vv[t1].r[2]};
        const float b[3] = {vv[t2].r[0], vv[t2].r[1], vv[t2].r[2]};
        const float c[3] = {vv[t3].r[0], vv[t3].r[1], vv[t3].r[2]};
            
        if (in_tetrahedron(p.r, a, b, c, origin)) ++count;
    }

    // dont consider the case of inside several solids
    if (count % 2) atomicExch(tags + gid, sid);
}
}
    
void inside_dev(const Particle *pp, const int n, const Mesh m, const Particle *i_pp, const int ns, /**/ int *tags) {
    if (ns == 0 || n == 0) return;
        
    kernels::init_tags <<< k_cnf(n) >>> (n, /**/ tags);

    dim3 thrd(128, 1);
    dim3 blck((127 + n)/128, ns);

    kernels::compute_tags <<< blck, thrd >>> (pp, n, i_pp, m.nv, m.tt, m.nt, /**/ tags);
}

static void get_bbox(const Particle *pp, const int n, /**/ float3 *minbb, float3 *maxbb) {
    if (n == 0) return;

    const float *r = pp[0].r;

    float3 minb = make_float3(r[0], r[1], r[2]);
    float3 maxb = make_float3(r[0], r[1], r[2]);

    auto min = [](float a, float b) {return a > b ? b : a;};
    auto max = [](float a, float b) {return a > b ? a : b;};
    
    for (int i = 1; i < n; ++i) {
        r = pp[i].r;
        minb.x = min(minb.x, r[0]); maxb.x = max(maxb.x, r[0]);
        minb.y = min(minb.y, r[1]); maxb.y = max(maxb.y, r[1]);
        minb.z = min(minb.z, r[2]); maxb.z = max(maxb.z, r[2]);
    }
    *minbb = minb; *maxbb = maxb;
}

void get_bbox(const float *rr, const int n, /**/ float3 *minbb, float3 *maxbb) {
    if (n == 0) return;

    const float *r = rr;
        
    float3 minb = make_float3(r[0], r[1], r[2]);
    float3 maxb = make_float3(r[0], r[1], r[2]);

    auto min = [](float a, float b) {return a > b ? b : a;};
    auto max = [](float a, float b) {return a > b ? a : b;};

    for (int i = 1; i < n; ++i) {
        r = rr + 3 * i;
        minb.x = min(minb.x, r[0]); maxb.x = max(maxb.x, r[0]);
        minb.y = min(minb.y, r[1]); maxb.y = max(maxb.y, r[1]);
        minb.z = min(minb.z, r[2]); maxb.z = max(maxb.z, r[2]);
    }
}

void get_bboxes_hst(const Particle *pp, const int nps, const int ns, /**/ float3 *minbb, float3 *maxbb) {
    for (int i = 0; i < ns; ++i)
    get_bbox(pp + i*nps, nps, /**/ minbb + i, maxbb + i);
}

void get_bboxes_dev(const Particle *pp, const int nps, const int ns, /**/ float3 *minbb, float3 *maxbb) {
    if (ns == 0) return;
    minmax(pp, nps, ns, /**/ minbb, maxbb);
}
}
