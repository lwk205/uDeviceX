#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>
#include <string.h>
#include <conf.h>
#include "inc/conf.h"

#include "d/api.h"
#include "utils/msg.h"
#include "utils/error.h"
#include "utils/imp.h"
#include "utils/cc.h"

#include "mpi/glb.h"
#include "mpi/wrapper.h"
#include "inc/dev.h"
#include "inc/type.h"
#include "parser/imp.h"
#include "partlist/type.h"
#include "clist/imp.h"
#include "pair/imp.h"

#include "coords/ini.h"
#include "coords/imp.h"

#include "cloud/imp.h"
#include "flu/type.h"
#include "fluforces/imp.h"

#include "io/txt/imp.h"

static Particle *pp, *pp0, *pp_hst;
static Force *ff, *ff_hst;
static int n;
static Clist clist;
static ClistMap *cmap;
static FluForcesBulk *bulkforces;

static void read_pp(const char *fname) {
    TxtRead *tr;
    size_t szp, szf;
    UC(txt_read_pp(fname, &tr));
    n = txt_read_get_n(tr);
    msg_print("have read %d particles", n);

    szp = (n + 32) * sizeof(Particle);
    szf = (n + 32) * sizeof(Force);

    UC(emalloc(szp, (void**)&pp_hst));
    UC(emalloc(szf, (void**)&ff_hst));

    CC(d::Malloc((void**)&pp, szp));
    CC(d::Malloc((void**)&pp0, szp));
    CC(d::Malloc((void**)&ff, szf));

    memcpy(pp_hst, txt_read_get_pp(tr), szp);
    CC(d::Memcpy(pp, pp_hst, szp, H2D));
    CC(d::Memset(ff, 0, szf));

    UC(txt_read_fin(tr));
}

static void dealloc() {
    CC(d::Free(pp));
    CC(d::Free(pp0));
    CC(d::Free(ff));
    UC(efree(pp_hst));
    UC(efree(ff_hst));
    n = 0;
}

static void build_clist() {
    UC(clist_build(n, n, pp, /**/ pp0, &clist, cmap));
    Particle *tmp = pp;
    pp = pp0;
    pp0 = tmp;
}

static void set_params(float dt, PairParams *p) {
    enum {ncolors = 2};
    float a[] = {adpd_b, adpd_br, adpd_r};
    float g[] = {gdpd_b, gdpd_br, gdpd_r};
    UC(pair_set_dpd(ncolors, a, g, p));
    UC(pair_compute_dpd_sigma(kBT, dt, /**/ p));
    UC(pair_set_lj(ljsigma, ljepsilon, p));
}

int main(int argc, char **argv) {
    Config *cfg;
    const char *fin, *fout;
    Cloud cloud;
    Coords *coords;
    int maxp;
    int3 L;
    PairParams *params;
    float dt;

    m::ini(&argc, &argv);
    msg_ini(m::rank);

    UC(conf_ini(&cfg));
    UC(conf_read(argc, argv, cfg));

    UC(coords_ini_conf(m::cart, cfg, &coords));
    L = subdomain(coords);

    UC(pair_ini(&params));
    UC(conf_lookup_float(cfg, "time.dt", &dt));
    UC(set_params(dt, params));

    UC(conf_lookup_string(cfg, "in", &fin));
    UC(conf_lookup_string(cfg, "out", &fout));
    UC(read_pp(fin));

    maxp = n + 32;

    UC(clist_ini(L.x, L.y, L.z, &clist));
    UC(clist_ini_map(maxp, 1, &clist, &cmap));
    UC(build_clist());

    UC(fluforces_bulk_ini(L, maxp, &bulkforces));

    UC(ini_cloud(pp, &cloud));

    UC(fluforces_bulk_prepare(n, &cloud, /**/ bulkforces));
    UC(fluforces_bulk_apply(params, n, bulkforces, clist.starts, clist.counts, /**/ ff));

    // particles are reordered because of clists
    CC(d::Memcpy(pp_hst, pp, n*sizeof(Particle), D2H));
    CC(d::Memcpy(ff_hst, ff, n*sizeof(Force)   , D2H));
    UC(txt_write_pp_ff(n, pp_hst, ff_hst, fout));

    UC(fluforces_bulk_fin(bulkforces));
    UC(clist_fin(&clist));
    UC(clist_fin_map(cmap));
    UC(dealloc());

    UC(pair_fin(params));
    UC(coords_fin(coords));
    UC(conf_fin(cfg));
    m::fin();
}
