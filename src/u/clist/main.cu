#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"

#include "msg.h"
#include "mpi/glb.h" /* mini-MPI and -device */
#include "d/api.h"

#include "glb.h"

#include "inc/dev.h"
#include "inc/type.h"
#include "utils/cc.h"

#include "algo/scan/int.h"
#include "clistx/imp.h"

enum {X,Y,Z};

#define MAXN 10000

void ini_1ppc(int3 d, int *n, Particle *pp) {
    int i, ix, iy, iz;
    Particle p;
    *n = d.x * d.y * d.z;

    for (i = 0, iz = 0; iz < d.z; ++iz)
        for (iy = 0; iy < d.y; ++iy)
            for (ix = 0; ix < d.x; ++ix) {
                p.r[X] = -d.x * 0.5f + ix + 0.5f;
                p.r[Y] = -d.y * 0.5f + iy + 0.5f;
                p.r[Z] = -d.z * 0.5f + iz + 0.5f;
                p.v[X] = p.v[Y] = p.v[Z] = 0.f;
                pp[i++] = p;
            }
}

void ini_random(int3 d, int density, int *n, Particle *pp) {
    Particle p;
    int i, N;
    N = *n = d.x * d.y * d.z * density;

    for (i = 0; i < N; ++i) {
        p.r[X] = (-0.5 + 0.999 * drand48()) * d.x;
        p.r[Y] = (-0.5 + 0.999 * drand48()) * d.y;
        p.r[Z] = (-0.5 + 0.999 * drand48()) * d.z;
        p.v[X] = p.v[Y] = p.v[Z] = 0.f;
        pp[i] = p;
    }
}

int3 ccoords(int3 d, int cid) {
    int3 c;
    c.x = cid % d.x;
    c.z = cid / (d.y * d.x);
    c.y = (cid - d.y * d.x * c.z) / d.x;
    return c;
}

bool valid(int c, int d, float x) {
    return (x >= c - 0.5 * d) && (x < c + 1 - 0.5 * d);
}

void verify_cell(int3 d, int cid, int s, int c, const Particle *pp) {
    int i, j;
    Particle p;
    int3 cell = ccoords(d, cid);
    for (i = 0; i < c; ++i) {
        j = s + i;
        p = pp[j];
        // MSG("%3f %3f %3f at %d %d %d",
        //     p.r[X], p.r[Y], p.r[Z], cell.x, cell.y, cell.z);
        assert(valid(cell.x, d.x, p.r[X]));
        assert(valid(cell.y, d.y, p.r[Y]));
        assert(valid(cell.z, d.z, p.r[Z]));
    }
}

void verify(int3 d, const int *starts, const int *counts, const Particle *pp, int n) {
    int cid, s, c, nc;
    nc = d.x * d.y * d.z;
    for (cid = 0; cid < nc; ++cid) {
        s = starts[cid];
        c = counts[cid];
        verify_cell(d, cid, s, c, pp);
    }
}

int main(int argc, char **argv) {
    m::ini(argc, argv);

    Particle *pplo, *ppre, *ppout;
    Particle *pp_hst;
    int nlo = 0, nre = 0, *starts, *counts, n;
    int3 dims = make_int3(4, 8, 4);
    clist::Clist clist;
    clist::Work work;

    ini(dims.x, dims.y, dims.z, /**/ &clist);
    ini_work(&clist, /**/ &work);

    pp_hst = (Particle*) malloc(MAXN * sizeof(Particle));
    counts = (int*) malloc(clist.ncells * sizeof(int));
    starts = (int*) malloc(clist.ncells * sizeof(int));
    CC(d::Malloc((void**) &pplo, MAXN * sizeof(Particle)));
    CC(d::Malloc((void**) &ppre, MAXN * sizeof(Particle)));
    CC(d::Malloc((void**) &ppout, MAXN * sizeof(Particle)));
       
    ini_1ppc(dims, /**/ &nlo, pp_hst);
    CC(d::Memcpy(pplo, pp_hst, nlo * sizeof(Particle), H2D));
    ini_random(dims, 4, /**/ &nre, pp_hst);
    CC(d::Memcpy(ppre, pp_hst, nre * sizeof(Particle), H2D));

    n = nlo + nre;
    
    build(nlo, nre, n, pplo, ppre, /**/ ppout, &clist, /*w*/ &work);
    
    CC(d::Memcpy(counts, clist.counts, clist.ncells * sizeof(int), D2H));
    CC(d::Memcpy(starts, clist.starts, clist.ncells * sizeof(int), D2H));
    CC(d::Memcpy(pp_hst, ppout, n * sizeof(Particle), D2H));
    
    verify(dims, starts, counts, pp_hst, n);    

    CC(d::Free(pplo));
    CC(d::Free(ppre));
    CC(d::Free(ppout));
    free(counts);
    free(starts);
    free(pp_hst);

    fin(/**/ &clist);
    fin_work(/**/ &work);

    
    m::fin();
}
