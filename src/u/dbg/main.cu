#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>

#include <conf.h>
#include "inc/conf.h"

#include "utils/msg.h"
#include "mpi/wrapper.h" /* mini-MPI and -device */
#include "mpi/glb.h"

#include "d/api.h"

#include "utils/cc.h"
#include "utils/kl.h"
#include "inc/type.h"
#include "inc/dev.h"
#include "dbg/imp.h"


const int n = 100;
Particle *pp;
Force *ff;

void alloc() {
    CC(d::Malloc((void**) &pp, n * sizeof(Particle)));
    CC(d::Malloc((void**) &ff, n * sizeof(Force)));
}

void free() {
    CC(d::Free(pp));
    CC(d::Free(ff));
}

namespace dev {

__global__ void fill_bugs(Particle *pp, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    Particle p;
    p.r[0] = p.r[1] = p.r[2] = 0;
    p.v[0] = p.v[1] = p.v[2] = 0;

    if (i >= n) return;
    if (i < 1) p.r[0] = 1.5 * XS;
    if (i < 1) p.v[0] = 0.f / 0.f; // nan
    pp[i] = p;
}

__global__ void fill_bugs(Force *ff, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    Force f;
    f.f[0] = f.f[1] = f.f[2] = 0;

    if (i >= n) return;
    if (i < 1) f.f[0] = 1.f / 0.f; // inf
    ff[i] = f;
}
} // dev

void fill_bugs() {
    KL(dev::fill_bugs, (k_cnf(n)), (pp, n));
    KL(dev::fill_bugs, (k_cnf(n)), (ff, n));
}

void check() {
    dbg::check_pos(pp, n, __FILE__, __LINE__, "pos");
    dbg::check_vv (pp, n, __FILE__, __LINE__, "vel");
    dbg::check_ff (ff, n, __FILE__, __LINE__, "acc");
}

int main(int argc, char **argv) {
    m::ini(&argc, &argv);
    alloc();
    fill_bugs();
    check();
    free();
    m::fin();
}
