#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>

#include <conf.h>
#include "inc/conf.h"

#include "utils/msg.h"
#include "mpi/wrapper.h" /* mini-MPI and -device */
#include "mpi/glb.h"

#include "d/api.h"

#include "utils/error.h"
#include "utils/cc.h"
#include "utils/kl.h"
#include "parser/imp.h"
#include "inc/type.h"
#include "inc/dev.h"
#include "dbg/imp.h"
#include "glob/type.h"
#include "glob/ini.h"

const int n = 100;
Particle *pp;
Force *ff;

void alloc() {
    CC(d::Malloc((void**) &pp, n * sizeof(Particle)));
    CC(d::Malloc((void**) &ff, n * sizeof(Force)));
}

void free() {
    CC(d::Free(pp));
    CC(d::Free(ff));
}

namespace dev {

__global__ void fill_bugs(Particle *pp, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    Particle p;
    p.r[0] = p.r[1] = p.r[2] = 0;
    p.v[0] = p.v[1] = p.v[2] = 0;

    if (i >= n) return;
    if (i < 1) p.r[0] = 1.5 * XS;
    if (i < 1) p.v[0] = 0.f / 0.f; // nan
    pp[i] = p;
}

__global__ void fill_bugs(Force *ff, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    Force f;
    f.f[0] = f.f[1] = f.f[2] = 0;

    if (i >= n) return;
    if (i < 1) f.f[0] = 1.f / 0.f; // inf
    ff[i] = f;
}
} // dev

void fill_bugs() {
    KL(dev::fill_bugs, (k_cnf(n)), (pp, n));
    KL(dev::fill_bugs, (k_cnf(n)), (ff, n));
}

void check(Coords c, Dbg *dbg) {
    UC(dbg_check_pos    (c, dbg, n, pp));
    UC(dbg_check_vel    (c, dbg, n, pp));
    UC(dbg_check_forces (c, dbg, n, ff));
}

int main(int argc, char **argv) {
    Dbg *dbg;
    Config *cfg;
    Coords coords;
    m::ini(&argc, &argv);
    UC(coords_ini(m::cart, &coords));
    
    UC(conf_ini(&cfg));
    UC(dbg_ini(&dbg));
    UC(conf_read(argc, argv, cfg));
    UC(dbg_set_conf(cfg, dbg));
    
    alloc();
    fill_bugs();
    check(coords, dbg);
    free();
    UC(dbg_fin(dbg));
    UC(conf_fin(cfg));
    UC(coords_fin(&coords));
    m::fin();
}
