
#include <hip/hip_runtime.h>
namespace edg {
void set(int md, int f, int x, int y,  /**/ int *hx, int *hy) {
    int j = f*md;
    while (hx[j] != -1) j++;
    hx[j] = x; hy[j] = y;
}

int get(int md, int i, int x, int *hx, int *hy) { /* next */
    i *= md;
    while (hx[i] != x) i++;
    return hy[i];
}
}
