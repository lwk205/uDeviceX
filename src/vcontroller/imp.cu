#include "hip/hip_runtime.h"
#include <stdio.h>
#include <mpi.h>
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"

#include "inc/type.h"
#include "inc/dev.h"
#include "d/api.h"
#include "mpi/wrapper.h"
#include "utils/cc.h"
#include "utils/kl.h"
#include "utils/mc.h"

#include "msg.h"

#include "math/dev.h"

#include "imp.h"
#include "dev.h"

static void reini_sampler(/**/ PidVCont *c) {
    int3 L = c->L;
    int ncells = L.x * L.y * L.z;

    if (ncells) CC(d::MemsetAsync(c->gridvel, 0, ncells * sizeof(float3)));

    c->nsamples = 0;
}

void ini(MPI_Comm comm, int3 L, float3 vtarget, float factor, /**/ PidVCont *c) {
    int ncells, nchunks;
    c->L = L;
    c->target = vtarget;
    c->current = make_float3(0, 0, 0);
    c->factor = factor;
    c->Kp = 2;
    c->Ki = 1;
    c->Kd = 8;

    MC(m::Comm_dup(comm, &c->comm));

    ncells = L.x * L.y * L.z;
    CC(d::Malloc((void **) &c->gridvel, ncells * sizeof(float3)));

    nchunks = ceiln(ncells, 32);
    
    CC(d::alloc_pinned((void **) &c->avgvel, nchunks * sizeof(float3)));
    CC(d::HostGetDevicePointer((void **) &c->davgvel, c->avgvel, 0));

    c->f = c->sume = make_float3(0, 0, 0);
    c->olde = vtarget;

    MC(m::Allreduce(&ncells, &c->totncells, 1, MPI_INT, MPI_SUM, c->comm));
    
    reini_sampler(/**/ c);
}

void fin(/**/ PidVCont *c) {
    CC(d::Free(c->gridvel));
    CC(d::FreeHost(c->avgvel));
    MC(m::Comm_free(&c->comm));
}

void sample(int n, const Particle *pp, const int *starts, const int *counts, /**/ PidVCont *c) {
    int3 L = c->L;
    
    dim3 block(8, 8, 1);
    dim3 grid(ceiln(L.x, block.x),
              ceiln(L.y, block.y),
              ceiln(L.z, block.z));

    KL(dev::sample, (grid, block), (L, starts, counts, (float2 *) pp, /**/ c->gridvel));
    
    c->nsamples ++;
}

float3 adjustF(/**/ PidVCont *c) {
    int3 L = c->L;
    int ncells, nchunks;
    ncells = L.x * L.y * L.z;
    nchunks = ceiln(ncells, 32);

    KL(dev::reduceByWarp, (nchunks, 32), (c->gridvel, ncells, /**/ c->davgvel));
    dSync();

    float3 vcur = make_float3(0, 0, 0), e, de;

    for (int i = 0; i < nchunks; ++i)
        add(c->avgvel + i, /**/ &vcur);

    MC(m::Allreduce(MPI_IN_PLACE, &vcur.x, 3, MPI_FLOAT, MPI_SUM, c->comm));

    const float fac = 1.0 / (c->totncells * c->nsamples);
    
    scal(fac, /**/ &vcur);

    c->current = vcur;

    diff(&c->target, &vcur, /**/ &e);
    diff(&e, &c->olde, /**/ &de);
    add(&e, /**/ &c->sume);

    c->f = make_float3(0, 0, 0);

    axpy(c->factor * c->Kp, &e,       /**/ &c->f);
    axpy(c->factor * c->Ki, &c->sume, /**/ &c->f);
    axpy(c->factor * c->Kd, &de,      /**/ &c->f);

    reini_sampler(/**/c);

    c->olde = e;
    return c->f;
}

void log(const PidVCont *c) {
    float3 v = c->current;
    float3 f = c->f;
    MSG("vcont [v] [f] :\n\t[ % .3e % .3e % .3e ]\n\t[ % .3e % .3e % .3e ]",
        v.x, v.y, v.z, f.x, f.y, f.z);
}
