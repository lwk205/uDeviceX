
#include "int.h"
#include "imp.h"

namespace wall {

void alloc_quants(Quants *q) {
    q->n = 0;
    q->pp = NULL;
}

void alloc_ticket(Ticket *t) {
    t->rnd   = new rnd::KISS;
    t->cells = new clist::Clist(XS + 2 * XWM, YS + 2 * YWM, ZS + 2 * ZWM);
}

void free_quants(Quants *q) {
    if (q->pp) CC(hipFree(q->pp));
    q->n = 0;
}

void free_ticket(Ticket *t) {
    delete t->cells;
    delete t->rnd;
    t->texstart.destroy();
    t->texpp.destroy();
}

void gen_quants(const sdf::Quants qsdf, /**/ int *n, Particle* pp, Quants *q) {
    sub::gen_quants(qsdf.texsdf, n, pp, &q->n, &q->pp);
}

void strt_quants(Quants *q) {
    sub::strt_quants(&q->n, &q->pp);
}

void gen_ticket(const Quants q, Ticket *t) {
    sub::gen_ticket(q.n, q.pp, t->cells, &t->texstart, &t->texpp);
}

void interactions(const sdf::Quants qsdf, const Quants q, const Ticket t, const int type, const Particle *pp, const int n, Force *ff) {
    sub::interactions(qsdf.texsdf, type, pp, n, t.texstart, t.texpp, q.n, /**/ t.rnd, ff);
}

void strt_dump_templ(const Quants q) {
    sub::strt_dump_templ(q.n, q.pp);
}

}
