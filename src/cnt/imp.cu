#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdint.h>
#include <conf.h>
#include "inc/conf.h"

#include "d/api.h"
#include "d/q.h"
#include "d/ker.h"

#include "algo/scan/imp.h"

#include "rnd/imp.h"
#include "rnd/dev.h"

#include "inc/def.h"
#include "msg.h"
#include "mpi/glb.h"
#include "utils/cc.h"

#include "frag/imp.h"

#include "utils/kl.h"
#include "inc/type.h"
#include "inc/dev.h"

#include "dbg/imp.h"

#include "inc/dev/common.h"

#include "forces/type.h"
#include "forces/use.h"
#include "forces/pack.h"
#include "forces/imp.h"

#include "clist/imp.h"
#include "clist/code.h"

#include "imp.h"

namespace cnt {

typedef Sarray<const float2*, MAX_OBJ_TYPES> float2pWraps;
typedef Sarray<      float *, MAX_OBJ_TYPES>  ForcepWraps;

enum {
    XOFFSET = XS / 2,
    YOFFSET = YS / 2,
    ZOFFSET = ZS / 2
};

namespace dev {
#include "dev/pair.h"
#include "dev/map/common.h"
#include "dev/map/halo.h"
#include "dev/map/bulk.h"
#include "dev/bulk.h"
#include "dev/halo.h"
}

#include "imp/bulk.h"
#include "imp/halo.h"
#include "imp/main.h"

} /* namespace */
