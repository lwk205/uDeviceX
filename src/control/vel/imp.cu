#include <stdio.h>
#include <mpi.h>
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"
#include "msg.h"

#include "utils/error.h"
#include "utils/efopen.h"

#include "inc/type.h"
#include "inc/dev.h"
#include "d/api.h"
#include "mpi/wrapper.h"
#include "mpi/glb.h"
#include "utils/cc.h"
#include "utils/kl.h"
#include "utils/mc.h"

#include "math/dev.h"


#include "imp.h"

namespace dev {
#include "dev/common.h"

#if   defined(VCON_CART)
#include "dev/cart.h"
#elif defined(VCON_RAD)
#include "dev/radial.h"
#else
#error VCON_* transformation undefined
#endif

#include "dev/sample.h"
} // dev

#include "imp/main.h"
