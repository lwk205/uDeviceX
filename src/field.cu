#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conf.h>

#include "inc/type.h"
#include "io/field.h"
#include "m.h"
#include "field.h"
#include "msg.h"

namespace field {
static float spl(float x) { /* b-spline (see tools/bspline.mac) */
    return  \
        x <= 0 ? 0.0 :
        x <= 1 ? x*x*x/6 :
        x <= 2 ? (x*((12-3*x)*x-12)+4)/6 :
        x <= 3 ? (x*(x*(3*x-24)+60)-44)/6 :
        x <= 4 ? (x*((12-x)*x-48)+64)/6 :
        0.0;
}

static void skip_line(FILE *f) {
    char l[BUFSIZ];
    fgets(l, sizeof(l), f);
}

static FILE* safe_fopen(const char *path, const char *mode) {
    FILE *f;
    f = fopen(path, mode);
    if (f == NULL) ERR("fail to open: %s\n", path);
    return f;
}

void ini_dims(const char *path, /**/ int N[3], float ext[3]) {
    FILE *f;
    char l[BUFSIZ];
    f = safe_fopen(path, "r");
    fgets(l, sizeof(l), f);
    sscanf(l, "%f %f %f", &ext[0], &ext[1], &ext[2]);
    fgets(l, sizeof(l), f);
    sscanf(l, "%d %d %d", &N[0], &N[1], &N[2]);
    fclose(f);
}
  
void ini_data(const char *path, int n, /**/ float *D) { /* read sdf file */
    FILE *f;
    f = safe_fopen(path, "r");
    skip_line(f); skip_line(f);
    fread(D, sizeof(D[0]), n, f);
    fclose(f);
}

void sample(const float org[3], const float spa[3], const int N0[3], const float *D0, const int N1[3], float *D1) {
    /* org: origin, spa: spacing, N[01]: number of points; D[01]: data
       sample from grid `0' to `1'
       org, spa: are for `0'
    */
    enum {X, Y, Z};
#define OOO(ix, iy, iz) (D1 [ix + N1[X] * (iy + N1[Y] * iz)])
#define DDD(ix, iy, iz) (D0 [ix + N0[X] * (iy + N0[Y] * iz)])
#define i2r(i, d) (org[d] + (i + 0.5) * spa[d] - 0.5)
#define i2x(i)    i2r(i,X)
#define i2y(i)    i2r(i,Y)
#define i2z(i)    i2r(i,Z)
    int iz, iy, ix, i, c, sx, sy, sz;
    float s;
    for (iz = 0; iz < N1[Z]; ++iz)
        for (iy = 0; iy < N1[Y]; ++iy)
            for (ix = 0; ix < N1[X]; ++ix) {
                float r[3] = {(float) i2x(ix), (float) i2y(iy), (float) i2z(iz)};

                int anchor[3];
                for (c = 0; c < 3; ++c) anchor[c] = (int)floor(r[c]);

                float w[3][4];
                for (c = 0; c < 3; ++c)
                    for (i = 0; i < 4; ++i)
                        w[c][i] = spl(r[c] - (anchor[c] - 1 + i) + 2);

                float tmp[4][4];
                for (sz = 0; sz < 4; ++sz)
                    for (sy = 0; sy < 4; ++sy) {
                        s = 0;
                        for (sx = 0; sx < 4; ++sx) {
                            int l[3] = {sx, sy, sz};
                            int g[3];
                            for (c = 0; c < 3; ++c)
                                g[c] = (l[c] - 1 + anchor[c] + N0[c]) % N0[c];

                            s += w[0][sx] * DDD(g[X], g[Y], g[Z]);
                        }
                        tmp[sz][sy] = s;
                    }
                float partial[4];
                for (sz = 0; sz < 4; ++sz) {
                    s = 0;
                    for (sy = 0; sy < 4; ++sy) s += w[1][sy] * tmp[sz][sy];
                    partial[sz] = s;
                }
                float val = 0;
                for (sz = 0; sz < 4; ++sz) val += w[2][sz] * partial[sz];
                OOO(ix, iy, iz) = val;
            }
#undef DDD
#undef OOO
}

static void scale0(int N[3], float s, int x, int y, int z, /**/ float *D) {
    enum {X, Y};
    int i;
    i = x + N[X] * (y + N[Y] * z);
    D[i] *= s;
}

void scale(int N[3], float s, /**/ float *D) {
    enum {X, Y, Z};
    int x, y, z;
    for (z = 0; z < N[Z]; ++z)
        for (y = 0; y < N[Y]; ++y)
            for (x = 0; x < N[X]; ++x)
                scale0(N, s, x, y, z, /**/ D);
}

static void dump0(const int N0[3], const float* D0, /**/ float* D1) {
    float org[3], spa[3];
    int c;
    float G; /* domain size ([g]lobal) */
    float lo; /* left edge of subdomain */
    int L[3] = {XS, YS, ZS};
    int *N1;

    N1 = L;
    for (c = 0; c < 3; ++c) {
        G = m::dims[c] * L[c];
        lo = m::coords[c] * L[c];
        spa[c] = N0[c] / G ;
        org[c] = lo / G * N0[c];
    }
    sample(org, spa, N0, D0,   N1, /**/ D1);
}

static void dump1(const int N[3], const float* D, /*w*/ float* W) {
    dump0(N, D, /**/ W);
    h5::scalar(W, "wall");
}

void dump(const int N[], const float* D) {
    float *W = new float[XS * YS * ZS];
    dump1(N, D, /*w*/ W);
    delete[] W;
}
} /* namespace field */
