#include "hip/hip_runtime.h"
#include <mpi.h>
#include "common.h"
#include "io.h"
#include <conf.h>
#include "m.h"
#include "l/m.h"
#include "field.h"

namespace field {
float spl(float x) { /* b-spline (see tools/bspline.mac) */
  return  \
    x <= 0 ? 0.0 :
    x <= 1 ? x*x*x/6 :
    x <= 2 ? (x*((12-3*x)*x-12)+4)/6 :
    x <= 3 ? (x*(x*(3*x-24)+60)-44)/6 :
    x <= 4 ? (x*((12-x)*x-48)+64)/6 :
    0.0;
}

void ini_dims(const char *path, int N[3], float extent[3]) {
    FILE *fh = fopen(path, "r");
    char line[2048];
    fgets(line, sizeof(line), fh);
    sscanf(line, "%f %f %f", &extent[0], &extent[1], &extent[2]);
    fgets(line, sizeof(line), fh);
    sscanf(line, "%d %d %d", &N[0], &N[1], &N[2]);
    fclose(fh);
}

void ini_data(const char *path, const int n, float *grid_data) { /* read sdf file */
    FILE *fh = fopen(path, "r");
    char line[2048];
    fgets(line, sizeof(line), fh);
    fgets(line, sizeof(line), fh);

    fread(grid_data, sizeof(float), n, fh);
    fclose(fh);
}

void sample(const float rlo[3], const float dr[3], const int nsize[3], const int N[3], const float ampl, const float *grid_data, float *out) {
    enum {X, Y, Z};
#define OOO(ix, iy, iz) (       out[ix + nsize[X] * (iy + nsize[Y] * iz)])
#define DDD(ix, iy, iz) (grid_data [ix +     N[X] * (iy +     N[Y] * iz)])
#define i2r(i, d) (rlo[d] + (i + 0.5) * dr[d] - 0.5)
#define i2x(i)    i2r(i,X)
#define i2y(i)    i2r(i,Y)
#define i2z(i)    i2r(i,Z)
    int iz, iy, ix, i, c, sx, sy, sz;
    float s;
    for (iz = 0; iz < nsize[Z]; ++iz)
    for (iy = 0; iy < nsize[Y]; ++iy)
    for (ix = 0; ix < nsize[X]; ++ix) {
        float r[3] = {(float) i2x(ix), (float) i2y(iy), (float) i2z(iz)};

        int anchor[3];
        for (c = 0; c < 3; ++c) anchor[c] = (int)floor(r[c]);

        float w[3][4];
        for (c = 0; c < 3; ++c)
        for (i = 0; i < 4; ++i)
	  w[c][i] = spl(r[c] - (anchor[c] - 1 + i) + 2);

        float tmp[4][4];
        for (sz = 0; sz < 4; ++sz)
        for (sy = 0; sy < 4; ++sy) {
            s = 0;
            for (sx = 0; sx < 4; ++sx) {
                int l[3] = {sx, sy, sz};
                int g[3];
                for (c = 0; c < 3; ++c)
                g[c] = (l[c] - 1 + anchor[c] + N[c]) % N[c];

                s += w[0][sx] * DDD(g[X], g[Y], g[Z]);
            }
            tmp[sz][sy] = s;
        }
        float partial[4];
        for (sz = 0; sz < 4; ++sz) {
            s = 0;
            for (sy = 0; sy < 4; ++sy) s += w[1][sy] * tmp[sz][sy];
            partial[sz] = s;
        }
        float val = 0;
        for (sz = 0; sz < 4; ++sz) val += w[2][sz] * partial[sz];
        OOO(ix, iy, iz) = val * ampl;
    }
#undef DDD
#undef OOO
}

void dump0(const int N[3], const float extent[3], const float* grid_data, float* walldata) {
    int c, L[3] = {XS, YS, ZS};
    float rlo[3], dr[3], ampl;
    for (c = 0; c < 3; ++c) {
        rlo[c] = m::coords[c] * L[c] / (float)(m::dims[c] * L[c]) * N[c];
        dr[c] = N[c] / (float)(m::dims[c] * L[c]);
    }
    ampl = L[0] / (extent[0] / (float) m::dims[0]);
    sample(rlo, dr, L, N, ampl, grid_data, walldata);
    H5FieldDump dump;
    dump.dump_scalarfield(walldata, "wall");
}

void dump(const int N[], const float extent[], const float* grid_data) {
    float *walldata = new float[XS * YS * ZS];
    dump0(N, extent, grid_data, walldata);
    delete[] walldata;
}
} /* namespace field */
