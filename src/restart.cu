#include <mpi.h>
#include "common.h"
#include <conf.h>
#include "conf.common.h"
#include "m.h"

#include "restart.h"

//#define DBG(...) MSG(__VA_ARGS__)
#define DBG(...) 

namespace restart {
enum {X, Y, Z};

/* pattern : 
   sing processor  : base/strt/code/id.ext
   mult processors : base/strt/code/XXX.YYY.ZZZ/id.ext
   base depends on read/write
 */
#define PF    "%s.%s"
#define DIR_S "%s/strt/%s/"                 PF
#define DIR_M "%s/strt/%s/%03d.%03d.%03d/"  PF

#define READ (true)
#define DUMP (false)

// buff size
#define BS (256)

// check fprintf (BS-1 for \0 character)
#define CSPR(a) do {                                        \
        int check = a;                                      \
        if (check < 0 || check >= BS-1)                     \
        ERR("Buffer too small to handle this format\n");    \
    } while (0)

#define CF(f, fname) do {if (f==NULL) ERR("could not open the file <%s>\n", fname);} while(0)

void id2str(const int id, char *str) {
    switch (id) {
    case TEMPL:
        CSPR(sprintf(str, "templ"));
        break;
    case FINAL:
        CSPR(sprintf(str, "final"));
        break;
    default:
        CSPR(sprintf(str, "%05d", id));
        break;
    }
}

void gen_name(const bool read, const char *code, const int id, const char *ext, /**/ char *name) {
    char idcode[BS] = {0};
    id2str(id, /**/ idcode);
    
    if (m::size == 1)
    CSPR(sprintf(name, DIR_S, read ? BASE_STRT_READ : BASE_STRT_DUMP, code, idcode, ext));
    else
    CSPR(sprintf(name, DIR_M, read ? BASE_STRT_READ : BASE_STRT_DUMP, code, m::coords[X], m::coords[Y], m::coords[Z], idcode, ext));
}

namespace bopwrite {
void header(const char *bop, const char *rel, const long n) {
    FILE *f = fopen(bop, "w"); CF(f, bop);
    
    fprintf(f, "%ld\n", n);
    fprintf(f, "DATA_FILE: %s\n", rel);
    fprintf(f, "DATA_FORMAT: float\n");
    fprintf(f, "VARIABLES: x y z vx vy vz\n");
    fclose(f);
}

template <typename T>
void data(const char *val, const T *dat, const long n) {
    FILE *f = fopen(val, "w"); CF(f, val);
    fwrite(dat, sizeof(T), n, f);
    fclose(f);
}
} // namespace bopwrite

namespace bopread {
void header(const char *name, long *n) {
    FILE *f = fopen(name, "r"); CF(f, name);
    if (fscanf(f, "%ld\n", n) != 1) ERR("wrong format\n");
    fclose(f);
}

template <typename T>
void data(const char *name, const long n, T *dat) {
    FILE *f = fopen(name, "r"); CF(f, name);
    fread(dat, sizeof(T), n, f);
    fclose(f);
}
} // namespace bopread

void write_pp(const char *code, const int id, const Particle *pp, const long n) {
    char bop[BS] = {0}, rel[BS] = {0}, val[BS] = {0}, idcode[BS] = {0};
    gen_name(DUMP, code, id, "bop"   , /**/ bop);
    gen_name(DUMP, code, id, "values", /**/ val);

    id2str(id, /**/ idcode);
    CSPR(sprintf(rel, PF, idcode, "values"));    

    bopwrite::header(bop, rel, n);
    bopwrite::data(val, pp, n);
}

void read_pp(const char *code, const int id, Particle *pp, int *n) {
    long np = 0;
    char bop[BS] = {0}, val[BS] = {0};
    gen_name(READ, code, id, "bop"   , /**/ bop);
    gen_name(READ, code, id, "values", /**/ val);
    DBG("reading <%s> and <%s>", bop, val);
    bopread::header(bop, &np);
    bopread::data(val, np, pp);
    *n = np;
    DBG("I have read %ld pp", np);
}

void write_ss(const char *code, const int id, const Solid *ss, const long n) {
    char fname[BS] = {0};
    gen_name(DUMP, code, id, "solid", /**/ fname);
        
    FILE *f = fopen(fname, "w"); CF(f, fname);
    fprintf(f, "%ld\n", n);
    fwrite(ss, sizeof(Solid), n, f);
    fclose(f);
}

void read_ss(const char *code, const int id, Solid *ss, int *n) {
    long ns = 0;
    char fname[BS] = {0};
    gen_name(READ, code, id, "solid", /**/ fname);
    fprintf(stderr, "reading %s\n", fname);
    FILE *f = fopen(fname, "r"); CF(f, fname);
    fscanf(f, "%ld\n", &ns);
    fread(ss, sizeof(Solid), ns, f);
    fclose(f);
    *n = ns;
    DBG("I have read %ld ss.", ns);
}

#undef PF
#undef DIR_S
#undef DIR_M

#undef READ
#undef DUMP

#undef DBG
} // namespace restart
