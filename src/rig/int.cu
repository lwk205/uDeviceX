#include "hip/hip_runtime.h"
#include <cstdio>

#include "common.h"
#include "common.cuda.h"

#include <conf.h>

#include "rig/int.h"
#include "rig/imp.h"

namespace rig {

void alloc_quants(Quants *q) {
    q->n = q->ns = q->nps = 0;
    
    CC(hipMalloc(&q->pp ,     MAX_PART_NUM * sizeof(Particle)));
    CC(hipMalloc(&q->ss ,       MAX_SOLIDS * sizeof(Solid)));
    CC(hipMalloc(&q->rr0, 3 * MAX_PART_NUM * sizeof(float)));
    CC(hipMalloc(&q->i_pp,    MAX_PART_NUM * sizeof(Particle)));
    
    q->pp_hst   = new Particle[MAX_PART_NUM];
    q->ss_hst   = new Solid[MAX_SOLIDS];
    q->rr0_hst  = new float[3 * MAX_PART_NUM];
    q->i_pp_hst = new Particle[MAX_PART_NUM];

    q->ss_dmp = new Solid[MAX_SOLIDS];
}

void free_quants(Quants *q) {
    delete[] q->pp_hst;
    delete[] q->ss_hst;
    delete[] q->rr0_hst;
    delete[] q->i_pp_hst;
    
    CC(hipFree(q->pp));
    CC(hipFree(q->ss));
    CC(hipFree(q->rr0));
    CC(hipFree(q->i_pp));

    if (q->m_hst.tt) delete[] q->m_hst.tt;
    if (q->m_hst.vv) delete[] q->m_hst.vv;

    if (q->m_dev.tt) CC(hipFree(q->m_dev.tt));
    if (q->m_dev.vv) CC(hipFree(q->m_dev.vv));

    delete[] q->ss_dmp;
}

void alloc_ticket(TicketBB *t) {
    CC(hipMalloc(&t->minbb_dev, MAX_SOLIDS * sizeof(float3)));
    CC(hipMalloc(&t->maxbb_dev, MAX_SOLIDS * sizeof(float3)));
    CC(hipMalloc(&t->i_pp,  MAX_PART_NUM * sizeof(Particle)));
    CC(hipMalloc(&t->ss ,        MAX_SOLIDS * sizeof(Solid)));

    // assume 1 triangle don't overlap more than 27 cells
    CC(hipMalloc(&t->tcs_dev, XS * YS * ZS * sizeof(int)));
    CC(hipMalloc(&t->tcc_dev, XS * YS * ZS * sizeof(int)));
    CC(hipMalloc(&t->tci_dev, 27 * MAX_SOLIDS * MAX_FACE_NUM * sizeof(int)));
    
    t->minbb_hst = new float3[MAX_SOLIDS];
    t->maxbb_hst = new float3[MAX_SOLIDS];
    t->ss_hst   = new Solid[MAX_SOLIDS];
    t->i_pp_hst = new Particle[MAX_PART_NUM];

    t->tcs_hst = new int[XS * YS * ZS];
    t->tcc_hst = new int[XS * YS * ZS];
    t->tci_hst = new int[27 * MAX_SOLIDS * MAX_FACE_NUM];

    t->ss_dmp = new Solid[MAX_SOLIDS];
}

void free_ticket(TicketBB *t) {
    CC(hipFree(t->minbb_dev));
    CC(hipFree(t->maxbb_dev));
    CC(hipFree(t->i_pp));
    CC(hipFree(t->ss));

    CC(hipFree(t->tcs_dev));
    CC(hipFree(t->tcc_dev));
    CC(hipFree(t->tci_dev));

    delete[] t->minbb_hst;
    delete[] t->maxbb_hst;
    delete[] t->ss_hst;
    delete[] t->i_pp_hst;

    delete[] t->tcs_hst;
    delete[] t->tcc_hst;
    delete[] t->tci_hst;

    delete[] t->ss_dmp;
}

static void cpy_H2D(Quants q) {
    cH2D(q.i_pp, q.i_pp_hst, q.ns * q.m_hst.nv);
    cH2D(q.ss,   q.ss_hst,   q.ns);
    cH2D(q.rr0,  q.rr0_hst,  q.nps * 3);
    cH2D(q.pp,   q.pp_hst,   q.n);
}

void gen_quants(Particle *opp, int *on, Quants *q) {
    sub::load_solid_mesh("mesh_solid.ply", /**/ &q->m_dev, &q->m_hst);
    sub::gen_from_solvent(q->m_hst, /**/ opp, on, /**/ &q->ns, &q->nps, &q->n, q->rr0_hst, q->ss_hst, q->pp_hst);
    sub::gen_pp_hst(q->ns, q->rr0_hst, q->nps, /**/ q->ss_hst, q->pp_hst);
    sub::gen_ipp_hst(q->ss_hst, q->ns, q->m_hst, /**/ q->i_pp_hst);
    cpy_H2D(*q);
}

void strt_quants(const int id, Quants *q) {
    sub::load_solid_mesh("mesh_solid.ply", /**/ &q->m_dev, &q->m_hst);
    sub::gen_from_strt(id, /**/ &q->ns, &q->nps, &q->n, q->rr0_hst, q->ss_hst);
    sub::gen_pp_hst(q->ns, q->rr0_hst, q->nps, /**/ q->ss_hst, q->pp_hst);
    sub::gen_ipp_hst(q->ss_hst, q->ns, q->m_hst, /**/ q->i_pp_hst);
    cpy_H2D(*q);
}

void set_ids(Quants q) {
    sub::set_ids(q.ns, q.ss_hst, q.ss);
}

void strt_dump_templ(const Quants q) {
    sub::strt_dump_templ(q.nps, q.rr0_hst);
}

void strt_dump(const int id, const Quants q) {
    sub::strt_dump(id, q.ns, q.ss_hst);
}

} // rig
