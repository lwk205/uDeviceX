#include <mpi.h>
#include <cassert>
#include <vector>

#include <conf.h>
#include "inc/conf.h"

#include "d/ker.h"
#include "d/api.h"

#include "m.h"
#include "inc/def.h"
#include "msg.h"
#include "cc.h"
#include "mc.h"

#include "mpi/wrapper.h"
#include "inc/type.h"
#include "inc/dev.h"
#include "texo.h"
#include "inc/mpi.type.h"
#include "rigid/int.h"

#include "mesh/collision.h"
#include "mesh/dist.h"
#include "mesh/bbox.h"

#include "io/ply.h"
#include "io/restart.h"

#include "rig/imp.h"
#include "rig/ic.h"
#include "rig/share.h"
#include "rig/ini.h"

namespace rig {
namespace sub {

void load_solid_mesh(const char *fname, Mesh *dev, Mesh *hst) {
    ply::read(fname, /**/ hst);

    dev->nv = hst->nv;
    dev->nt = hst->nt;

    CC(hipMalloc(&dev->tt, 3 * dev->nt * sizeof(int)));
    CC(hipMalloc(&dev->vv, 3 * dev->nv * sizeof(float)));

    cH2D(dev->tt, hst->tt, 3 * dev->nt);
    cH2D(dev->vv, hst->vv, 3 * dev->nv);
}

void gen_from_solvent(const Mesh m_hst,  /* io */ Particle *opp, int *on,
                      /* o */ int *ns, int *nps, int *n, float *rr0_hst, Solid *ss_hst, Particle *pp_hst) {
    // generate models
    MSG("start solid ini");
    ic::ini("rigs-ic.txt", m_hst, /**/ ns, nps, rr0_hst, ss_hst, on, opp, pp_hst);
    MSG("done solid ini");

    *n = *ns * (*nps);
}

static void pp2rr(const Particle *pp, const int n, float *rr) {
    for (int i = 0; i < n; ++i)
    for (int c = 0; c < 3; ++c)
    rr[3*i + c] = pp[i].r[c];
}

void gen_from_strt(const int id, int *ns, int *nps, int *n, float *rr0_hst, Solid *ss_hst) {
    Particle *pp = new Particle[MAX_PART_NUM];
    restart::read_pp("rig", restart::TEMPL, pp, nps);
    pp2rr(pp, *nps, rr0_hst);
    delete[] pp;

    restart::read_ss("rig", id, ss_hst, ns);
    *n = *ns * (*nps);
}

void gen_pp_hst(const int ns, const float *rr0_hst, const int nps, /**/ Solid *ss_hst, Particle *pp_hst) {
    rig::generate_hst(ss_hst, ns, rr0_hst, nps, /**/ pp_hst);
    rig::reinit_ft_hst(ns, /**/ ss_hst);
}

void gen_ipp_hst(const Solid *ss_hst, const int ns, const Mesh m_hst, Particle *i_pp_hst) {
    rig::mesh2pp_hst(ss_hst, ns, m_hst, /**/ i_pp_hst);
}

void set_ids(const int ns, Solid *ss_hst, Solid *ss_dev) {
    ic::set_ids(ns, ss_hst);
    if (ns) cH2D(ss_dev, ss_hst, ns);
}

static void rr2pp(const float *rr, const int n, Particle *pp) {
    for (int i = 0; i < n; ++i)
    for (int c = 0; c < 3; ++c) {
        pp[i].r[c] = rr[3*i + c];
        pp[i].v[c] = 0;
    }
}

void strt_dump_templ(const int nps, const float *rr0_hst) {
    Particle *pp = new Particle[nps];
    rr2pp(rr0_hst, nps, pp);

    restart::write_pp("rig", restart::TEMPL, pp, nps);
    
    delete[] pp;
}

void strt_dump(const int id, const int ns, const Solid *ss) {
    restart::write_ss("rig", id, ss, ns);
}

} // sub
} // rig
