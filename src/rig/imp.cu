#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cassert>
#include <vector>

#include <conf.h>
#include "inc/conf.h"

#include "d/ker.h"
#include "d/api.h"

#include "mpi/glb.h"
#include "inc/def.h"
#include "msg.h"
#include "utils/cc.h"
#include "utils/mc.h"

#include "mpi/wrapper.h"
#include "inc/type.h"
#include "inc/dev.h"
#include "utils/texo.h"
#include "mpi/type.h"
#include "rigid/int.h"

#include "mesh/collision.h"
#include "mesh/dist.h"
#include "mesh/bbox.h"

#include "io/ply.h"
#include "io/restart.h"

#include "rig/imp.h"
#include "rig/ic.h"
#include "rig/share.h"
#include "rig/ini.h"

namespace rig {
namespace sub {

void load_solid_mesh(const char *fname, int *nt, int *nv, int4 **tt_hst, int4 **tt_dev, float **vv_hst, float **vv_dev) {
    ply::read(fname, /**/ nt, nv, tt_hst, vv_hst);

    CC(hipMalloc(tt_dev, *nt * sizeof(int4)));
    CC(hipMalloc(vv_dev, 3 * *nv * sizeof(float)));

    cH2D(*tt_dev, *tt_hst, *nt);
    cH2D(*vv_dev, *vv_hst, 3 * *nv);
}

void gen_from_solvent(int nt, int nv, const int4 *tt, const float *vv, /* io */ Particle *opp, int *on,
                      /* o */ int *ns, int *nps, int *n, float *rr0_hst, Solid *ss_hst, Particle *pp_hst) {
    // generate models
    MSG("start solid ini");
    ic::ini("rigs-ic.txt", nt, nv, tt, vv, /**/ ns, nps, rr0_hst, ss_hst, on, opp, pp_hst);
    MSG("done solid ini");

    *n = *ns * (*nps);
}

static void pp2rr(const Particle *pp, const int n, float *rr) {
    for (int i = 0; i < n; ++i)
    for (int c = 0; c < 3; ++c)
    rr[3*i + c] = pp[i].r[c];
}

void gen_from_strt(const int id, int *ns, int *nps, int *n, float *rr0_hst, Solid *ss_hst) {
    Particle *pp = new Particle[MAX_PART_NUM];
    restart::read_pp("rig", restart::TEMPL, pp, nps);
    pp2rr(pp, *nps, rr0_hst);
    delete[] pp;

    restart::read_ss("rig", id, ss_hst, ns);
    *n = *ns * (*nps);
}

void gen_pp_hst(const int ns, const float *rr0_hst, const int nps, /**/ Solid *ss_hst, Particle *pp_hst) {
    rig::generate_hst(ss_hst, ns, rr0_hst, nps, /**/ pp_hst);
    rig::reinit_ft_hst(ns, /**/ ss_hst);
}

void gen_ipp_hst(const Solid *ss_hst, const int ns, int nv, const float *vv, Particle *i_pp_hst) {
    rig::mesh2pp_hst(ss_hst, ns, nv, vv, /**/ i_pp_hst);
}

void set_ids(const int ns, Solid *ss_hst, Solid *ss_dev) {
    ic::set_ids(ns, ss_hst);
    if (ns) cH2D(ss_dev, ss_hst, ns);
}

static void rr2pp(const float *rr, const int n, Particle *pp) {
    for (int i = 0; i < n; ++i)
    for (int c = 0; c < 3; ++c) {
        pp[i].r[c] = rr[3*i + c];
        pp[i].v[c] = 0;
    }
}

void strt_dump_templ(const int nps, const float *rr0_hst) {
    Particle *pp = new Particle[nps];
    rr2pp(rr0_hst, nps, pp);

    restart::write_pp("rig", restart::TEMPL, pp, nps);
    
    delete[] pp;
}

void strt_dump(const int id, const int ns, const Solid *ss) {
    restart::write_ss("rig", id, ss, ns);
}

} // sub
} // rig
