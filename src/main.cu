#include <cstdio>
#include <mpi.h>
#include <map>
#include ".conf.h" /* configuration file (copy from .conf.test.h) */
#include "m.h"     /* MPI */
#include "common.h"
#include "bund.h"
#include "glb.h"

void mpi_init(int argc, char **argv) {
  MC(MPI_Init(&argc, &argv));
  MC(MPI_Comm_rank(MPI_COMM_WORLD,   &m::rank));
  MC(MPI_Cart_create(MPI_COMM_WORLD,
		     m::d, m::dims, m::periods, m::reorder,   &m::cart));
  MC(MPI_Cart_coords(m::cart, m::rank, m::d,   m::coords));
}

int main(int argc, char **argv) {
  m::dims[0] = m::dims[1] = m::dims[2] = 1;
  for (int iarg = 1; iarg < argc && iarg <= 3; iarg++)
    m::dims[iarg - 1] = atoi(argv[iarg]);

  int device = 2;
  CC(hipSetDevice(device));

  mpi_init(argc, argv);
  glb::sim(); /* simulation level globals */

  sim::init();
  sim::run();
  sim::close();
  
  MC(MPI_Finalize());
}
