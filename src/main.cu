#include <cstdio>
#include <mpi.h>
#include <map>
#include <conf.h>
#include "m.h"     /* MPI */
#include "common.h"
#include "bund.h"
#include "glb.h"

void mpi_init(int argc, char **argv) {
    MC(MPI_Init(&argc, &argv));
    MC(l::m::Comm_rank(MPI_COMM_WORLD,   &m::rank));
    MC(l::m::Cart_create(MPI_COMM_WORLD,
                       m::d, m::dims, m::periods, m::reorder,   &m::cart));
    MC(l::m::Cart_coords(m::cart, m::rank, m::d,   m::coords));
}

int main(int argc, char **argv) {
    m::dims[0] = m::dims[1] = m::dims[2] = 1;
    for (int iarg = 1; iarg < argc && iarg <= 3; iarg++)
    m::dims[iarg - 1] = atoi(argv[iarg]);

    mpi_init(argc, argv);

    // panda specific for multi-gpu testing
    //int device = m::rank % 2 ? 0 : 2;
    int device = 0;
    CC(hipSetDevice(device));
  
    glb::sim(); /* simulation level globals */

    sim::init();
    sim::run();
    sim::close();
  
    MC(l::m::Finalize());
}
