#include "hip/hip_runtime.h"
#include <sys/stat.h> /* for dump.impl */

#include <assert.h>
#include <vector>
#include <mpi.h>

#include "l/m.h"
#include "l/off.h"
#include "scan/int.h"

#include <limits> /* for rnd */
#include <stdint.h>
#include "rnd.h"

#include <conf.h>
#include "conf.common.h"
#include "m.h"     /* MPI */
#include "basetags.h"
#include "common.h"
#include "inc/type.h"
#include "common.mpi.h"
#include "common.cuda.h"
#include "inc/tmp/wrap.h"
#include "inc/tmp/pinned.h"
#include "io.h"
#include "bund.h"
#include "diag.h"

#include "restart.h"

#include "glb.h"

#include "l/float3.h"
#include "k/read.h"
#include "k/write.h"
#include "k/common.h"

#include "clist/int.h"
#include "minmax.h"

namespace mcomm {
namespace sub {
namespace dev {
#include "mcomm/dev.h"
}
#include "mcomm/ini.h"
#include "mcomm/imp.h"
#include "mcomm/fin.h"
}
#include "mcomm/int.h"
}

namespace rbc {
namespace sub {
#define __DF__ __device__ __forceinline__
/* physical part of RBC calculations : see also params/rbc.inc0.h */
#include "rbc/dev0.h"
#include "rbc/dev.h"
#undef __DF__
#include "rbc/imp.h"
}
namespace ic {
#include "rbc/ic.h"
}
#include "rbc/int.h"
}

#include "mdstr/buf.h"
#include "mdstr/tic.h"
#include "mdstr/int.h"
#include "rdstr/int.h"

#include "sdstr.decl.h"
#include "sdstr.impl.h"
#include "field.h"

#include "forces.h"

#include "k/cnt/type.h"
#include "k/cnt/decl.h"
#include "k/cnt/bulk.h"
#include "k/cnt/halo.h"
#include "k/cnt/pop.h"
#include "cnt/decl.h"
#include "cnt/bind.h"
#include "cnt/build.h"
#include "cnt/bulk.h"
#include "cnt/fin.h"
#include "cnt/halo.h"
#include "cnt/ini.h"
#include "cnt/setup.h"

#include "sdf/type.h"
#include "sdf/int.h"

#include "wall/int.h"

#include "flu/int.h"

#include "odstr/type.h"
#include "odstr/int.h"

#include "k/fsi/type.h"
#include "k/fsi/decl.h"
#include "k/fsi/common.h"
#include "k/fsi/map.common.h"
#include "k/fsi/map.bulk.h"
#include "k/fsi/map.halo.h"
#include "k/fsi/bulk.h"
#include "k/fsi/halo.h"

#include "fsi/decl.h"
#include "fsi/setup.h"
#include "fsi/bind.h"
#include "fsi/bulk.h"
#include "fsi/fin.h"
#include "fsi/halo.h"
#include "fsi/ini.h"

#include "x/type.h"
  #include "rex/type.h"
  #include "rex/decl.h"

  #include "k/rex/type.h"
  #include "k/rex/decl.h"
  #include "k/rex/common.h"
  #include "k/rex/x.h" /* TODO */
  #include "k/rex/pack.h"
  #include "k/rex/scan.h"
  #include "k/rex/scatter.h"
  #include "k/rex/unpack.h"

  #include "rex/ini.h"
  #include "rex/copy.h"
  #include "rex/wait.h"
  #include "rex/halo.h"
  #include "rex/scan.h"
  #include "rex/pack.h"
  #include "rex/send.h"
  #include "rex/recv.h"
  #include "rex/unpack.h"
  #include "rex/post.h"
  #include "rex/fin.h"

#include "x/decl.h"
#include "x/common.h"
#include "x/ticketcom.h"
#include "x/ticketr.h"
#include "x/tickettags.h"
#include "x/ticketpack.h"
#include "x/ticketpinned.h"
#include "x/impl.h"

#include "dpd/local.h"
#include "dpd/flocal.h"

namespace dpdx {
namespace dev {
#include "dpd/x/dev.h"
}
#include "dpd/x/imp.h"
}

#include "dpdr/type.h"
#include "dpdr/int.h"

#include "mesh/collision.h"
#include "mesh/bbox.h"

#include "solid.h"
#include "tcells.h"

#include "mbounce/imp.h"
#include "mrescue.h"

#include "bbhalo.decl.h"
#include "bbhalo.impl.h"

#include "dump.h"

#include "rig/int.h"

namespace sim {
namespace dev {
#include "sim/dev.h"
}
#include "sim/dec.h"
#include "sim/ini.h"
#include "sim/fin.h"
#include "sim/generic.h"
#include "sim/dump.h"
#include "sim/tag.h"
#include "sim/forces.h"
#define HST (true)
#define DEV (false)
#define DEVICE_SOLID
#ifdef DEVICE_SOLID
  #include "0dev/sim.impl.h"
#else
  #include "0hst/sim.impl.h"
#endif
#undef HST
#undef DEV
#include "sim/update.h"
#include "sim/step.h"
#include "sim/run.h"
#include "sim/imp.h"
}
