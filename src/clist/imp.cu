#include "hip/hip_runtime.h"
#include <cstdio>
#include <conf.h>
#include "conf.common.h"
#include "cc.h"
#include "kl.h"

#include "inc/type.h"
#include "common.h"
#include "common.cuda.h"
#include "scan/int.h"
#include "clist/int.h"

namespace clist {
namespace dev {
#include "clist/dev.h"
}

static void scan(const int *counts, int n, /**/ int *starts) {
    scan::Work ws;
    scan::alloc_work(n, /**/ &ws);
    scan::scan(counts, n, /**/ starts, /*w*/ &ws);
    scan::free_work(&ws);
}

void build(int n, int xcells, int ycells, int zcells,
           float xstart, float ystart, float zstart,
           /**/ Particle *pp, int *starts, int *counts) {
    if (!n) return;

    const int ncells = xcells * ycells * zcells;
    if (!ncells) return;

    const int3 cells = make_int3(xcells, ycells, zcells);
    const int3 domainstart = make_int3(xstart, ystart, zstart);
    
    int *ids;
    Particle *ppd;
    CC(hipMalloc(&ids, n*sizeof(ids[0])));
    CC(hipMalloc(&ppd, n*sizeof(ppd[0])));

    CC(hipMemsetAsync(counts, 0, ncells * sizeof(int)));

    KL(dev::get_counts, (k_cnf(n)), (pp, n, cells, domainstart, /**/ counts));

    scan(counts, ncells, /**/ starts);
    
    CC(hipMemsetAsync(counts, 0, ncells * sizeof(int)));

    KL(dev::get_ids, (k_cnf(n)), (pp, starts, n, cells, domainstart, /**/ counts, ids));

    KL(dev::gather, (k_cnf(n)), (pp, ids, n, /**/ ppd));

    CC(hipMemcpyAsync(pp, ppd, n * sizeof(Particle), D2D));
    
    CC(hipFree(ids));
    CC(hipFree(ppd));
}
}
