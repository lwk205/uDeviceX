#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"
#include "inc/def.h"
#include "utils/error.h"
#include "utils/msg.h"
#include "utils/cc.h"

#include "d/q.h"
#include "d/ker.h"
#include "d/api.h"

#include "inc/type.h"
#include "inc/dev.h"
#include "utils/texo.h"
#include "utils/te.h"
#include "utils/texo.dev.h"

#include "utils/kl.h"
#include "mesh/collision.h"
#include "rigid/imp.h"

enum {OUT=BLUE_COLOR, IN=RED_COLOR};
enum {X, Y, Z};

static __host__ __device__ bool same_side(const float *x, const float *p, const float *a, const float *b, const float *inplane) {
    const float n[3] = {a[Y] * b[Z] - a[Z] * b[Y],
                        a[Z] * b[X] - a[X] * b[Z],
                        a[X] * b[Y] - a[Y] * b[X]};

    const float ndx = n[X] * (x[X] - inplane[X]) + n[Y] * (x[Y] - inplane[Y]) + n[Z] * (x[Z] - inplane[Z]);
    const float ndp = n[X] * (p[X] - inplane[X]) + n[Y] * (p[Y] - inplane[Y]) + n[Z] * (p[Z] - inplane[Z]);

    return ndx * ndp > 0;
}

static __host__ __device__ bool in_tetrahedron(const float *x, const float *A, const float *B, const float *C, const float *D) {
    const float AB[3] = {B[X] - A[X], B[Y] - A[Y], B[Z] - A[Z]};
    const float AC[3] = {C[X] - A[X], C[Y] - A[Y], C[Z] - A[Z]};
    const float AD[3] = {D[X] - A[X], D[Y] - A[Y], D[Z] - A[Z]};

    const float BC[3] = {C[X] - B[X], C[Y] - B[Y], C[Z] - B[Z]};
    const float BD[3] = {D[X] - B[X], D[Y] - B[Y], D[Z] - B[Z]};

    return
        same_side(x, A, BC, BD, D) &&
        same_side(x, B, AD, AC, D) &&
        same_side(x, C, AB, BD, D) &&
        same_side(x, D, AB, AC, A);
}

int collision_inside_1p(const RigPinInfo *pi, const float *r, const float *vv, const int4 *tt, const int nt) {
    int c = 0;
    float origin[3] = {0, 0, 0};
    int spdir = rig_get_pdir(pi);

    if (spdir != NOT_PERIODIC)
        origin[spdir] = r[spdir];

    for (int i = 0; i < nt; ++i) {
        int4 t = tt[i];
        if (in_tetrahedron(r, vv + 3*t.x, vv + 3*t.y, vv + 3*t.z, origin)) ++c;
    }
    return c%2;
}

static int inside_1p(int spdir, const float *r, const Particle *vv, const int4 *tt, const int nt) {
    int c = 0;
    float origin[3] = {0, 0, 0};

    if (spdir != NOT_PERIODIC)
        origin[spdir] = r[spdir];

    for (int i = 0; i < nt; ++i) {
        int4 t = tt[i];
        if (in_tetrahedron(r, vv[t.x].r, vv[t.y].r, vv[t.z].r, origin)) ++c;
    }
    return c%2;
}

void collision_inside_hst(int spdir, const Particle *pp, const int n, int nt, int nv, const int4 *tt, const Particle *i_pp, const int ns, /**/ int *tags) {
    for (int i = 0; i < n; ++i) {
        tags[i] = -1;
        for (int sid = 0; sid < ns; ++sid)
            if (inside_1p(spdir, pp[i].r, i_pp + nv * sid, tt, nt)) {
            tags[i] = sid;
            break;
        }
    }
}

namespace collisiondev
{
__global__ void init_tags(const int n, const int color, /**/ int *tags) {
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < n) tags[gid] = color;
}

/* assume ns blocks along y */
/* if the ith particle is inside jth mesh, sets tag[i] to j */
__global__ void compute_tags(int spdir, const Particle *pp, const int n, const Particle *vv, const int nv, const int4 *tt, const int nt, /**/ int *tags) {
    const int sid = blockIdx.y;
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;

    int count = 0;

    const Particle p = pp[gid];
    float origin[3] = {0, 0, 0};

    if (spdir != NOT_PERIODIC)
        origin[spdir] = p.r[spdir];

    for (int i = 0; i < nt; ++i) {
        int4 t = tt[i];
        const int t1 = sid * nv + t.x;
        const int t2 = sid * nv + t.y;
        const int t3 = sid * nv + t.z;

        const float a[3] = {vv[t1].r[0], vv[t1].r[1], vv[t1].r[2]};
        const float b[3] = {vv[t2].r[0], vv[t2].r[1], vv[t2].r[2]};
        const float c[3] = {vv[t3].r[0], vv[t3].r[1], vv[t3].r[2]};

        if (in_tetrahedron(p.r, a, b, c, origin)) ++count;
    }

    // dont consider the case of inside several solids
    if (count % 2) atomicExch(tags + gid, sid);
}

union Pos {
    float2 f2[2];
    struct { float r[3]; float dummy; };
};

__device__ Pos tex2Pos(const Texo<float2> texvert, const int id) {
    Pos r;
    r.f2[0] = fetch(texvert, 3 * id + 0);
    r.f2[1] = fetch(texvert, 3 * id + 1);
    return r;
}

static __device__ bool inside_box(const float r[3], float3 lo, float3 hi) {
    enum {X, Y, Z};
    return
        r[X] >= lo.x && r[X] <= hi.x &&
        r[Y] >= lo.y && r[Y] <= hi.y &&
        r[Z] >= lo.z && r[Z] <= hi.z;
}

/* assume nm blocks along y */
/* if the ith particle is inside jth mesh, sets tag[i] to IN (see enum in collision.h) */
__global__ void compute_colors_tex(int spdir, const Particle *pp, const int n, const Texo<float2> texvert, const int nv,
                                   const int4 *tri,
                                   const int nt, const float3 *minext, const float3 *maxext, /**/ int *cc) {
    const int sid = blockIdx.y;
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;

    int count = 0;

    const Particle p = pp[gid];

    float3 lo, hi;
    lo = minext[sid];
    hi = maxext[sid];
    if (!inside_box(p.r, lo, hi)) return;

    float origin[3] = {0, 0, 0};

    if (spdir != NOT_PERIODIC)
        origin[spdir] = p.r[spdir];

    int mbase = nv * sid;
    for (int i = 0; i < nt; ++i) {
        const int4 t = tri[i];

        const Pos a = tex2Pos(texvert, mbase + t.x);
        const Pos b = tex2Pos(texvert, mbase + t.y);
        const Pos c = tex2Pos(texvert, mbase + t.z);

        if (in_tetrahedron(p.r, a.r, b.r, c.r, origin)) ++count;
    }

    // dont consider the case of inside several solids
    if (count % 2) atomicExch(cc + gid, IN);
}
}

void collision_inside_dev(int spdir, const Particle *pp, const int n, int nt, int nv, const int4 *tt, const Particle *i_pp, const int ns, /**/ int *tags) {
    if (ns == 0 || n == 0) return;

    KL(collisiondev::init_tags, (k_cnf(n)), (n, -1, /**/ tags));

    enum {THR = 128};
    dim3 thrd(THR, 1);
    dim3 blck(ceiln(n, THR), ns);

    KL(collisiondev::compute_tags, (blck, thrd), (spdir, pp, n, i_pp, nv, tt, nt, /**/ tags));
}

/*
   n:  number of particles
   nm: number of meshes
   nt: number of triangles per mesh
   nv: number of vertices per mesh
*/
static void get_colors0(int spdir, const Particle *pp, int n,
                        const Texo<float2> texvert, const int4 *tri,
                        int nt, int nv, int nm,
                        const float3 *minext, const float3 *maxext, /**/ int *cc) {
    if (nm == 0 || n == 0) return;

    KL(collisiondev::init_tags, (k_cnf(n)), (n, OUT, /**/ cc));

    enum {THR = 128};
    dim3 thrd(THR, 1);
    dim3 blck(ceiln(n, THR), nm);

    KL(collisiondev::compute_colors_tex, (blck, thrd), (spdir, pp, n, texvert, nv, tri, nt, minext, maxext, /**/ cc));
}

void collision_get_colors(int spdir, const Particle *pp, int n,
                          const Particle *i_pp, const int4 *tri,
                          int nt, int nv, int nm,
                          const float3 *minext, const float3 *maxext, /**/ int *cc) {
    Texo<float2> texvert;
    if (nm == 0 || n == 0) return;
    TE(&texvert, (float2*) i_pp, 3 * nm * nv);
    UC(get_colors0(spdir, pp, n, texvert, tri,
                   nt, nv, nm,
                   minext, maxext, /**/ cc));
    destroy(&texvert);
}

