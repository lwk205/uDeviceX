#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"
#include "inc/def.h"
#include "utils/error.h"
#include "utils/msg.h"
#include "utils/cc.h"

#include "d/q.h"
#include "d/ker.h"
#include "d/api.h"

#include "inc/type.h"
#include "inc/dev.h"
#include "utils/texo.h"
#include "utils/texo.dev.h"
#include "mesh/triangles/type.h"
#include "mesh/triangles/imp.h"

#include "utils/kl.h"
#include "rigid/imp.h"

#include "imp.h"

enum {X, Y, Z};

static __host__ __device__ bool same_side(const float *x, const float *p, const float *a, const float *b, const float *inplane) {
    const float n[3] = {a[Y] * b[Z] - a[Z] * b[Y],
                        a[Z] * b[X] - a[X] * b[Z],
                        a[X] * b[Y] - a[Y] * b[X]};

    const float ndx = n[X] * (x[X] - inplane[X]) + n[Y] * (x[Y] - inplane[Y]) + n[Z] * (x[Z] - inplane[Z]);
    const float ndp = n[X] * (p[X] - inplane[X]) + n[Y] * (p[Y] - inplane[Y]) + n[Z] * (p[Z] - inplane[Z]);

    return ndx * ndp > 0;
}

static __host__ __device__ bool in_tetrahedron(const float *x, const float *A, const float *B, const float *C, const float *D) {
    const float AB[3] = {B[X] - A[X], B[Y] - A[Y], B[Z] - A[Z]};
    const float AC[3] = {C[X] - A[X], C[Y] - A[Y], C[Z] - A[Z]};
    const float AD[3] = {D[X] - A[X], D[Y] - A[Y], D[Z] - A[Z]};

    const float BC[3] = {C[X] - B[X], C[Y] - B[Y], C[Z] - B[Z]};
    const float BD[3] = {D[X] - B[X], D[Y] - B[Y], D[Z] - B[Z]};

    return
        same_side(x, A, BC, BD, D) &&
        same_side(x, B, AD, AC, D) &&
        same_side(x, C, AB, BD, D) &&
        same_side(x, D, AB, AC, A);
}

int collision_inside_1p(int spdir, const float *r, const float *vv, const int4 *tt, const int nt) {
    int c = 0;
    float origin[3] = {0, 0, 0};

    if (spdir != NOT_PERIODIC)
        origin[spdir] = r[spdir];

    for (int i = 0; i < nt; ++i) {
        int4 t = tt[i];
        if (in_tetrahedron(r, vv + 3*t.x, vv + 3*t.y, vv + 3*t.z, origin)) ++c;
    }
    return c%2;
}

namespace collision_dev
{
__global__ void init_tags(const int n, const int color, /**/ int *tags) {
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < n) tags[gid] = color;
}

union Pos {
    float2 f2[2];
    struct { float r[3]; float dummy; };
};

static __device__ Pos tex2Pos(const Texo<float2> texvert, const int id) {
    Pos r;
    r.f2[0] = texo_fetch(texvert, 3 * id + 0);
    r.f2[1] = texo_fetch(texvert, 3 * id + 1);
    return r;
}

static __device__ bool inside_box(const float r[3], float3 lo, float3 hi) {
    enum {X, Y, Z};
    return
        r[X] >= lo.x && r[X] <= hi.x &&
        r[Y] >= lo.y && r[Y] <= hi.y &&
        r[Z] >= lo.z && r[Z] <= hi.z;
}

/* assume nm blocks along y */
__global__ void label_tex(int pdir, const Particle *pp, const int n, const Texo<float2> texvert, const int nv,
                          Triangles tri, const float3 *minext, const float3 *maxext,
                          int lab_in, /**/ int *labels) {
    int i, sid, gid, count, mbase;
    Particle p;
    Pos a, b, c;
    float3 lo, hi;
    int4 t;
    sid = blockIdx.y;
    gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;

    count = 0;

    p = pp[gid];
    
    lo = minext[sid];
    hi = maxext[sid];
    if (!inside_box(p.r, lo, hi)) return;

    float origin[3] = {0, 0, 0};
    if (pdir != NOT_PERIODIC) origin[pdir] = p.r[pdir];

    mbase = nv * sid;
    for (i = 0; i < tri.nt; ++i) {
        t = tri.tt[i];

        a = tex2Pos(texvert, mbase + t.x);
        b = tex2Pos(texvert, mbase + t.y);
        c = tex2Pos(texvert, mbase + t.z);

        if (in_tetrahedron(p.r, a.r, b.r, c.r, origin)) ++count;
    }

    // dont consider the case of inside several solids
    if (count % 2) atomicExch(labels + gid, lab_in);
}
}

static void label(int pdir, int n, const Particle *pp, const Triangles *tri, int nv, int nm, const Texo<float2> texvert,                        
                       const float3 *minext, const float3 *maxext, int lab_in, int lab_out, /**/ int *labels) {
    enum {X, Y, Z};
    if (nm == 0 || n == 0) return;

    KL(collision_dev::init_tags, (k_cnf(n)), (n, lab_out, /**/ labels));

    enum {THR = 128};
    dim3 thrd(THR, 1);
    dim3 blck(ceiln(n, THR), nm);

    KL(collision_dev::label_tex, (blck, thrd),
       (pdir, pp, n, texvert, nv, *tri, minext, maxext, lab_in, /**/ labels)); 
}

void collision_label(int pdir, int n, const Particle *pp, const Triangles *tri, 
                     int nv, int nm, const Particle *i_pp, 
                     const float3 *minext, const float3 *maxext,
                     int lab_in, int lab_out, /**/ int *labels) {
    Texo<float2> texvert;
    int ntex;
    ntex = 3 * nm * nv;
    
    if (nm == 0 || n == 0) return;
    texo_setup(ntex, (float2*) i_pp, /**/ &texvert);
    UC(label(pdir, n, pp, tri, nv, nm, texvert, minext, maxext, lab_in, lab_out, /**/ labels));
    texo_destroy(&texvert);
}
