#include <stdio.h>

#include "conf.h"
#include "inc/conf.h"

#include "d/api.h"

#include "utils/imp.h"
#include "utils/error.h"
#include "utils/cc.h"

#include "sdf/array3d/type.h"

#include "type.h"
#include "imp.h"

void tex3d_ini(Tex3d **pq) {
    Tex3d *q;
    UC(emalloc(sizeof(Tex3d), /**/ (void**)&q));
    *pq = q;
}

void tex3d_fin(Tex3d *q) {
    CC(hipDestroyTextureObject(q->t));
    UC(efree(q));
}

void tex3d_copy(Array3d *a, /**/ Tex3d *t) {
    hipResourceDesc resD;
    hipTextureDesc  texD;

    memset(&resD, 0, sizeof(resD));
    resD.resType = hipResourceTypeArray;
    resD.res.array.array = a->a;

    memset(&texD, 0, sizeof(texD));
    texD.normalizedCoords = 0;
    texD.filterMode = hipFilterModePoint;
    texD.mipmapFilterMode = hipFilterModePoint;
    texD.addressMode[0] = hipAddressModeWrap;
    texD.addressMode[1] = hipAddressModeWrap;
    texD.addressMode[2] = hipAddressModeWrap;

    CC(hipCreateTextureObject(&t->t, &resD, &texD, NULL));
}
