#include <mpi.h>
#include <stdio.h>
#include <conf.h>
#include "inc/conf.h"

#include "d/ker.h"
#include "d/api.h"

#include "utils/error.h"
#include "inc/def.h"
#include "msg.h"
#include "mpi/glb.h"
#include "utils/cc.h"

#include "inc/type.h"
#include "inc/dev.h"
#include "inc/macro.h"

#include "sdf/type.h"
#include "sdf/int.h"
#include "sdf/imp.h"

namespace sdf {
void alloc_quants(Quants *q) {
    hipChannelFormatDesc fmt = hipCreateChannelDesc<float>();
    CC(hipMalloc3DArray(&q->arrsdf, &fmt, make_hipExtent(XTE, YTE, ZTE)));
}

void  free_quants(Quants *q) {
    CC(hipFreeArray(q->arrsdf));
    q->texsdf.destroy();
}

void ini(MPI_Comm cart, Quants *q) {
    UC(sub::ini(cart, q->arrsdf, &q->texsdf));
}

void bulk_wall(const tex3Dca<float> texsdf, /*io*/ Particle *s_pp, int *s_n, /*o*/ Particle *w_pp, int *w_n) {
    UC(sub::bulk_wall(texsdf, /*io*/ s_pp, s_n, /*o*/ w_pp, w_n));
}

int who_stays(const Quants q, Particle *pp, int n, int nc, int nv, int *stay) {
    return sub::who_stays(q.texsdf, pp, n, nc, nv, /**/ stay);
}

void bounce(const Quants *q, int n, /**/ Particle *pp) {
    UC(sub::bounce(q->texsdf, n, /**/ pp));
}
}
