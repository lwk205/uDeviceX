#include "common.h"

#include "sdf/type.h"
#include "sdf/int.h"
#include "sdf/imp.h"

namespace sdf {
void alloc_quants(Quants *q) {
    hipChannelFormatDesc fmt = hipCreateChannelDesc<float>();
    CC(hipMalloc3DArray(&q->arrsdf, &fmt, make_hipExtent(XTE, YTE, ZTE)));
}

void  free_quants(Quants *q) {
    CC(hipFreeArray(q->arrsdf));
    q->texsdf.destroy();
}

void ini(Quants *q) {
    sub::ini(q->arrsdf, &q->texsdf);
}

int who_stays(const Quants q, Particle *pp, int n, int nc, int nv, int *stay) {
    return sub::who_stays(q.texsdf, pp, n, nc, nv, /**/ stay);
}

void bounce(const tex3Dca<float> texsdf, int n, /**/ Particle *pp) {
    sub::bounce(texsdf, n, /**/ pp);
}
}
