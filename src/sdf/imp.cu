#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conf.h>
#include "inc/conf.h"

#include "mpi/glb.h"
#include "inc/def.h"
#include "msg.h"
#include "utils/cc.h"

#include "d/q.h"
#include "d/ker.h"
#include "d/api.h"

#include "inc/type.h"
#include "inc/dev.h"
#include "inc/macro.h"

#include "utils/kl.h"
#include "glb.h"
#include "inc/dev/wvel.h"

#include "field.h"

#include "sdf/type.h"
#include "sdf/imp.h"
#include "sdf/cheap.dev.h"
#include "sdf/dev.h"

namespace sdf {
namespace sub {

struct Tex { /* simplifies communication between ini[0123..] */
    hipArray *a;
    tex3Dca<float> *t;
};

struct Wa { /* local wall data */
    const tex3Dca<float> texsdf;
    float gd;
};

static void ini0(float *D, /**/ struct Tex te) {
    hipMemcpy3DParms copyParams;
    memset(&copyParams, 0, sizeof(copyParams));
    copyParams.srcPtr = make_hipPitchedPtr((void*)D, XTE * sizeof(float), XTE, YTE);
    copyParams.dstArray = te.a;
    copyParams.extent = make_hipExtent(XTE, YTE, ZTE);
    copyParams.kind = H2D;
    CC(hipMemcpy3D(&copyParams));
    te.t->setup(te.a);
}

static void ini1(int N[3], float *D0, float *D1, /**/ struct Tex te) {
    int c;
    int L[3] = {XS, YS, ZS};
    int M[3] = {XWM, YWM, ZWM}; /* margin and texture */
    int T[3] = {XTE, YTE, ZTE};
    float G; /* domain size ([g]lobal) */
    float lo; /* left edge of subdomain */
    float org[3], spa[3]; /* origin and spacing */
    for (c = 0; c < 3; ++c) {
        G = m::dims[c] * L[c];
        lo = m::coords[c] * L[c];
        spa[c] = N[c] * (L[c] + 2 * M[c]) / G / T[c];
        org[c] = N[c] * (lo - M[c]) / G;
    }
    field::sample(org, spa, N, D0,   T, /**/ D1);
    ini0(D1, te);
}

static void ini2(int N[3], float* D0, /**/ struct Tex te) {
    float *D1 = new float[XTE * YTE * ZTE];
    ini1(N, D0, D1, /**/ te);
    delete[] D1;
}

static void ini3(int N[3], float ext[3], float* D, /**/ struct Tex te) {
    enum {X, Y, Z};
    float sc, G; /* domain size in x ([G]lobal) */
    G = m::dims[X] * XS;
    sc = G / ext[X];
    field::scale(N, sc, /**/ D);

    /* MC(l::m::Barrier(l::m::cart)); */
    if (field_dumps) field::dump(N, D);

    ini2(N, D, /**/ te);
}

void ini(hipArray *arrsdf, tex3Dca<float> *texsdf) {
    enum {X, Y, Z};
    float *D;     /* data */
    int N[3];     /* size of D */
    float ext[3]; /* extent */
    int n;
    char f[] = "sdf.dat";
    struct Tex te {arrsdf, texsdf};

    field::ini_dims(f, /**/ N, ext);
    n = N[X] * N[Y] * N[Z];
    D = new float[n];
    field::ini_data(f, n, /**/ D);
    ini3(N, ext, D, /**/ te);
    delete[] D;
}

/* sort solvent particle (dev) into remaining in solvent (dev) and turning into wall (hst)*/
static void bulk_wall0(const tex3Dca<float> texsdf, /*io*/ Particle *s_pp, int* s_n,
                       /*o*/ Particle *w_pp, int *w_n, /*w*/ int *keys) {
    int n = *s_n;
    int k, a = 0, b = 0, w = 0; /* all, bulk, wall */
    KL(dev::fill,(k_cnf(n)), (texsdf, s_pp, n, keys));
    for (/* */ ; a < n; a++) {
        cD2H(&k, &keys[a], 1);
        if      (k == W_BULK) {cD2D(&s_pp[b], &s_pp[a], 1); b++;}
        else if (k == W_WALL) {cD2H(&w_pp[w], &s_pp[a], 1); w++;}
    }
    *s_n = b; *w_n = w;
}

void bulk_wall(const tex3Dca<float> texsdf, /*io*/ Particle *s_pp, int *s_n, /*o*/ Particle *w_pp, int *w_n) {
    int *keys;
    Dalloc(&keys, MAX_PART_NUM);
    bulk_wall0(texsdf, s_pp, s_n, w_pp, w_n, keys);
    CC(hipFree(keys));
}

/* bulk predicate : is in bulk? */
static bool bulkp(int *keys, int i) {
    int k; cD2H(&k, &keys[i], 1);
    return k == W_BULK;
}

static int who_stays0(int *keys, int nc, int nv, /*o*/ int *stay) {
    int c, v;  /* cell and vertex */
    int s = 0; /* how many stays? */
    for (c = 0; c < nc; ++c) {
        v = 0;
        while (v  < nv && bulkp(keys, v + nv * c)) v++;
        if    (v == nv) stay[s++] = c;
    }
    return s;
}

static int who_stays1(const tex3Dca<float> texsdf, Particle *pp, int n, int nc, int nv, /**/ int *stay, /*w*/ int *keys) {
    KL(dev::fill, (k_cnf(n)), (texsdf, pp, n, keys));
    return who_stays0(keys, nc, nv, /**/ stay);
}

int who_stays(const tex3Dca<float> texsdf, Particle *pp, int n, int nc, int nv, /**/ int *stay) {
    int *keys;
    CC(hipMalloc(&keys, n*sizeof(keys[0])));
    nc = who_stays1(texsdf, pp, n, nc, nv, /**/ stay, /*w*/ keys);
    CC(hipFree(keys));
    return nc;
}

void bounce(const tex3Dca<float> texsdf, int n, /**/ Particle *pp) {
    Wa wa;
    wa.texsdf = texsdf;
    wa.gd     = gamma_dot;

    KL(dev::bounce, (k_cnf(n)), (texsdf, n, /**/ (float2*) pp));
}

} // sub
} // sdf
