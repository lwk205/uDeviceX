#include <stdio.h>

#include "conf.h"
#include "inc/conf.h"

#include "d/api.h"

#include "utils/imp.h"
#include "utils/error.h"
#include "utils/cc.h"

#include "type.h"
#include "imp.h"

void array3d_ini(Array3d **pq, int x, int y, int z) {
    Array3d *q;
    hipChannelFormatDesc fmt;

    UC(emalloc(sizeof(Array3d), /**/ (void**)&q));

    fmt = hipCreateChannelDesc<float>();
    CC(hipMalloc3DArray(&q->a, &fmt, make_hipExtent(x, y, z)));
    q->x = x; q->y = y; q->z = z;

    *pq = q;
}

void array3d_fin(Array3d *q) {
    CC(hipFreeArray(q->a));
    UC(efree(q));
}

static int good(size_t x, size_t y, size_t z, Array3d *q) {
    return x == q->x && y == q->y && z == q->z;
}

void array3d_copy(int x, int y, int z, float *D, /**/ Array3d *q) {
    hipMemcpy3DParms copyParams;
    if (!good(x, y, z, q))
        ERR("wrong size: %ld, %ld, %ld   !=   %ld, %ld, %ld",
            x, y, z, q->x, q->y, q->z);
    memset(&copyParams, 0, sizeof(copyParams));
    copyParams.srcPtr = make_hipPitchedPtr((void*)D, x*sizeof(float), x, y);
    copyParams.dstArray = q->a;
    copyParams.extent = make_hipExtent(x, y, z);
    copyParams.kind = hipMemcpyHostToDevice;
    CC(hipMemcpy3D(&copyParams));
}
