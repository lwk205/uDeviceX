#include "hip/hip_runtime.h"
#include <mpi.h>
#include <utility>
#include <cstdio>
#include "conf.h"
#include "conf.common.h"
#include "m.h"     /* MPI */
#include "common.h"

bool Particle::initialized = false;
MPI_Datatype Particle::mytype;

bool Solid::initialized = false;
MPI_Datatype Solid::mytype;

void diagnostics(Particle * particles, int n, int idstep) {
    double p[] = {0, 0, 0};
    for(int i = 0; i < n; ++i)
    for(int c = 0; c < 3; ++c)
    p[c] += particles[i].v[c];

    MC(MPI_Reduce(m::rank == 0 ? MPI_IN_PLACE : &p,
                  m::rank == 0 ? &p : NULL, 3,
                  MPI_DOUBLE, MPI_SUM, 0, m::cart) );
    double ke = 0;
    for(int i = 0; i < n; ++i)
    ke += pow(particles[i].v[0], 2) + pow(particles[i].v[1], 2) + pow(particles[i].v[2], 2);

    MC(MPI_Reduce(m::rank == 0 ? MPI_IN_PLACE : &ke,
                  &ke,
                  1, MPI_DOUBLE, MPI_SUM, 0, m::cart));
    MC(MPI_Reduce(m::rank == 0 ? MPI_IN_PLACE : &n,
                  &n, 1, MPI_INT, MPI_SUM, 0, m::cart));

    double kbt = 0.5 * ke / (n * 3. / 2);
    if (m::rank == 0) {
        static bool firsttime = true;
        FILE * f = fopen("diag.txt", firsttime ? "w" : "a");
        firsttime = false;
        if (idstep == 0)
        fprintf(f, "# TSTEP\tKBT\tPX\tPY\tPZ\n");

        printf("\x1b[91m timestep: %e\t%.10e\t%.10e\t%.10e\t%.10e\x1b[0m\n", idstep * dt, kbt, p[0], p[1], p[2]);
        fprintf(f, "%e\t%.10e\t%.10e\t%.10e\t%.10e\n", idstep * dt, kbt, p[0], p[1], p[2]);
        fclose(f);
    }
}
