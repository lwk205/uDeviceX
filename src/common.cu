#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>
#include <conf.h>
#include "m.h"     /* MPI */
#include "common.h"

bool Particle::initialized = false;
MPI_Datatype Particle::mytype;

bool Solid::initialized = false;
MPI_Datatype Solid::mytype;

void diagnostics(Particle *pp, int n, int idstep) {
    double p[] = {0, 0, 0};
    for(int i = 0; i < n; ++i)
    for(int c = 0; c < 3; ++c)
    p[c] += pp[i].v[c];

    MC(l::m::Reduce(m::rank == 0 ? MPI_IN_PLACE : &p,
                  m::rank == 0 ? &p : NULL, 3,
                  MPI_DOUBLE, MPI_SUM, 0, m::cart) );
    double ke = 0;
    for(int i = 0; i < n; ++i)
    ke += pow(pp[i].v[0], 2) + pow(pp[i].v[1], 2) + pow(pp[i].v[2], 2);

    MC(l::m::Reduce(m::rank == 0 ? MPI_IN_PLACE : &ke,
                  &ke,
                  1, MPI_DOUBLE, MPI_SUM, 0, m::cart));
    MC(l::m::Reduce(m::rank == 0 ? MPI_IN_PLACE : &n,
                  &n, 1, MPI_INT, MPI_SUM, 0, m::cart));

    double kbt = 0.5 * ke / (n * 3. / 2);
    if (m::rank == 0) {
        static bool firsttime = true;
        FILE * f = fopen("diag.txt", firsttime ? "w" : "a");
        firsttime = false;
        if (idstep == 0) fprintf(f, "# TSTEP\tKBT\tPX\tPY\tPZ\n");
        fprintf(stderr, "%e\t%.10e\t%.10e\t%.10e\t%.10e\n", idstep * dt, kbt, p[0], p[1], p[2]);
        fprintf(f, "%e\t%.10e\t%.10e\t%.10e\t%.10e\n", idstep * dt, kbt, p[0], p[1], p[2]);
        fclose(f);
    }
}
