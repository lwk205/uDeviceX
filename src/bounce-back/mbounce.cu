#include "hip/hip_runtime.h"
#ifndef dt
#include "../.conf.h"
#endif
#include "../common.h"

#include <cassert>

#include "roots.h"
#include "mbounce.h"

namespace mbounce
{
    enum {X, Y, Z};

    enum BBState
    {
        BB_SUCCESS,   /* succesfully bounced            */
        BB_NOCROSS,   /* did not cross the plane        */
        BB_WTRIANGLE, /* [w]rong triangle               */
        BB_HFAIL      /* no time solution h             */
    };

#define _DH_ __device__ __host__

#define BBOX_MARGIN 0.1f

#define debug_output
    
    static void rvprev(const float *r1, const float *v1, const float *f0, /**/ float *r0, float *v0)
    {
#ifdef FORWARD_EULER
        for (int c = 0; c < 3; ++c)
        {
            v0[c] = v1[c] - f0[c] * dt;
            r0[c] = r1[c] - v0[c] * dt;
        }
#else // velocity-verlet
        for (int c = 0; c < 3; ++c)
        {
            r0[c] = r1[c] - v1[c] * dt;
            //v0[c] = v1[c] - f0[c] * dt;

            // BB assumes r0 + v0 dt = r1 for now
            v0[c] = v1[c];
        }
#endif
    }
    
    static bool cubic_root(float a, float b, float c, float d, /**/ float *h)
    {
        #define valid(t) ((t) >= 0 && (t) <= dt)
        #define eps 1e-8
        
        if (fabs(a) > eps) // cubic
        {
            const float sc = 1.f / a;
            b *= sc; c *= sc; d *= sc;
            
            float h1, h2, h3;
            int nsol = roots::cubic(b, c, d, &h1, &h2, &h3);

            if (valid(h1))             {*h = h1; return true;}
            if (nsol > 1 && valid(h2)) {*h = h2; return true;}
            if (nsol > 2 && valid(h3)) {*h = h3; return true;}
        }
        else if(fabs(b) > eps) // quadratic
        {
            float h1, h2;
            if (!roots::quadratic(b, c, d, &h1, &h2)) return false;
            if (valid(h1)) {*h = h1; return true;}
            if (valid(h2)) {*h = h2; return true;}
        }
        else if (fabs(c) < eps) // linear
        {
            const float h1 = -d/c;

            if (valid(h1)) {*h = h1; return true;}
        }
        return false;
    }
    
    /* see Fedosov PhD Thesis */
    static BBState intersect_triangle(const float *s10, const float *s20, const float *s30,
                                      const float *vs1, const float *vs2, const float *vs3,
                                      const Particle *p0,  /**/ float *h, float *rw)
    {
#define diff(a, b) {a[X] - b[X], a[Y] - b[Y], a[Z] - b[Z]}
#define cross(a, b) {a[Y] * b[Z] - a[Z] * b[Y], a[Z] * b[X] - a[X] * b[Z], a[X] * b[Y] - a[Y] * b[X]}
#define dot(a, b) (a[X]*b[X] + a[Y]*b[Y] + a[Z]*b[Z])
#define apxb(a, x, b) {a[X] + (float) x * b[X], a[Y] + (float) x * b[Y], a[Z] + (float) x * b[Z]} 
        
        const float *r0 = p0->r;
        const float *v0 = p0->v;
    
        const float a1[3] = diff(s20, s10);
        const float a2[3] = diff(s30, s10);
    
        const float at1[3] = diff(vs2, vs1);
        const float at2[3] = diff(vs3, vs1);

        // n(t) = n + t*nt + t^2 * ntt
        const float n0[3] = cross(a1, a2);
        const float ntt[3] = cross(at1, at2);
        const float nt[3] = {a1[Y] * at2[Z] - a1[Z] * at2[Y]  +  at1[Y] * a2[Z] - at1[Z] * a2[Y],
                             a1[Z] * at2[X] - a1[X] * at2[Z]  +  at1[Z] * a2[X] - at1[X] * a2[Z],
                             a1[X] * at2[Y] - a1[Y] * at2[X]  +  at1[X] * a2[Y] - at1[Y] * a2[X]};
    
        const float dr0[3] = diff(r0, s10);
        
        // check intersection with plane
        {
            const float r1[3] = apxb(r0, dt, v0);
            const float s11[3] = apxb(s10, dt, vs1);

            const float n1[3] = {n0[X] + (float) dt * (nt[X] + (float) dt * ntt[X]),
                                 n0[Y] + (float) dt * (nt[Y] + (float) dt * ntt[Y]),
                                 n0[Z] + (float) dt * (nt[Z] + (float) dt * ntt[Z])};
            
            const float dr1[3] = diff(r1, s11);

            const float b0 = dot(dr0, n0);
            const float b1 = dot(dr1, n1);

            if (b0 * b1 > 0)
            return BB_NOCROSS;
        }

        // find intersection time with plane

        const float dv[3] = diff(v0, vs1);
        
        const float a = dot(ntt, dv);
        const float b = dot(ntt, dr0) + dot(nt, dv);
        const float c = dot(nt, dr0) + dot(n0, dv);
        const float d = dot(n0, dr0);
        
        if (!cubic_root(a, b, c, d, h))
        return BB_HFAIL;

        rw[X] = r0[X] + *h * v0[X];
        rw[Y] = r0[Y] + *h * v0[Y];
        rw[Z] = r0[Z] + *h * v0[Z];

        // check if inside triangle

        {
            const float g[3] = {rw[X] - s10[X] - *h * vs1[X],
                                rw[Y] - s10[Y] - *h * vs1[Y],
                                rw[Z] - s10[Z] - *h * vs1[Z]};

            const float a1_[3] = apxb(a1, *h, at1);
            const float a2_[3] = apxb(a2, *h, at2);
            
            const float ga1 = dot(g, a1_);
            const float ga2 = dot(g, a2_);
            const float a11 = dot(a1_, a1_);
            const float a12 = dot(a1_, a2_);
            const float a22 = dot(a2_, a2_);

            const float fac = 1.f / (a11*a22 - a12*a12);
            
            const float u = (ga1 * a22 - ga2 * a12) * fac;
            const float v = (ga2 * a11 - ga1 * a12) * fac;

            if (!((u >= 0) && (v >= 0) && (u+v <= 1)))
            return BB_WTRIANGLE;
        }

        return BB_SUCCESS;
    }

    static _DH_ void lin_mom_solid(const float *v1, const float *vn, /**/ float *dP)
    {
        for (int c = 0; c < 3; ++c)
        dP[c] = -(vn[c] - v1[c]) / dt;
    }

    static _DH_ void ang_mom_solid(const float *com, const float *rw, const float *v0, const float *vn, /**/ float *dL)
    {
        const float dr[3] = {rw[X] - com[X], rw[Y] - com[Y], rw[Z] - com[Z]};
        
        dL[X] = -(dr[Y] * vn[Z] - dr[Z] * vn[Y] - dr[Y] * v0[Z] + dr[Z] * v0[Y]) / dt;
        dL[Y] = -(dr[Z] * vn[X] - dr[X] * vn[Z] - dr[Z] * v0[X] + dr[X] * v0[Z]) / dt;
        dL[Z] = -(dr[X] * vn[Y] - dr[Y] * vn[X] - dr[X] * v0[Y] + dr[Y] * v0[X]) / dt;
    }

#ifdef debug_output
    int bbstates[4], dstep = 0;
#endif
    
    static void bounce_1s1p(const float *f, const Mesh m, Particle *p, Solid *s)
    {
        float dL[3] = {0}, dP[3] = {0};

        const Particle p1 = *p;
        Particle p0;
        
        rvprev(p1.r, p1.v, f, /**/ p0.r, p0.v);

        for (int it = 0; it < m.nt; ++it)
        {
            const int t1 = m.tt[3*it + 0];
            const int t2 = m.tt[3*it + 1];
            const int t3 = m.tt[3*it + 2];

            float a1[3] = {m.vv[3*t1+0], m.vv[3*t1+1], m.vv[3*t1+2]};
            float a2[3] = {m.vv[3*t2+0], m.vv[3*t2+1], m.vv[3*t2+2]};
            float a3[3] = {m.vv[3*t3+0], m.vv[3*t3+1], m.vv[3*t3+2]};

            get_vl_solid(a1, vcml, oml, /**/ v1);
            get_vl_solid(a2, vcml, oml, /**/ v2);
            get_vl_solid(a3, vcml, oml, /**/ v3);

#define revert(a, v) do {                       \
                a[X] -= v[X] * dt;              \
                a[Y] -= v[Y] * dt;              \
                a[Z] -= v[Z] * dt;              \
            } while(0)

            revert(a1, v1);
            revert(a2, v2);
            revert(a3, v3);

            const BBState bbstate = intersect_triangle(a1, a2, a3, v1, v2, v3, &p0l, /**/ &h, rwl);

#ifdef debug_output
            bbstates[bbstate] ++;
#endif
            if (bbstate == BB_SUCCESS)
            {
                get_vl_solid(rwl, vcml, oml, /**/ vwl);

                pnl.v[X] = 2 * vwl[X] - p0l.v[X];
                pnl.v[Y] = 2 * vwl[Y] - p0l.v[Y];
                pnl.v[Z] = 2 * vwl[Z] - p0l.v[Z];

                pnl.r[X] = rwl[X] + (dt - h) * pnl.v[X];
                pnl.r[Y] = rwl[Y] + (dt - h) * pnl.v[Y];
                pnl.r[Z] = rwl[Z] + (dt - h) * pnl.v[Z];

                r2global(s->e0, s->e1, s->e2, s->com, rwl,   /**/ rw);
                r2global(s->e0, s->e1, s->e2, s->com, pnl.r, /**/ p->r);
                v2global(s->e0, s->e1, s->e2,         pnl.v, /**/ p->v);
                v2global(s->e0, s->e1, s->e2, p0l.v,         /**/ v0);
                
                lin_mom_solid(v0, p->v, /**/ dP);
                ang_mom_solid(s->com, rw, v0, p->v, /**/ dL);
                
                break;
            }
        }

        s->fo[X] += dP[X];
        s->fo[Y] += dP[Y];
        s->fo[Z] += dP[Z];

        s->to[X] += dL[X];
        s->to[Y] += dL[Y];
        s->to[Z] += dL[Z];
    }
    
    static bool in_bbox(const float *rg, const float *com, const float *bbox, const float tol)
    {
        const float rl[3] = {rg[X] - com[X], rg[Y] - com[Y], rg[Z] - com[Z]};
        
        return (rl[X] >= bbox[2*X + 0] - tol) && (rl[X] < bbox[2*X + 1] + tol) &&
            (rl[Y] >= bbox[2*Y + 0] - tol) && (rl[Y] < bbox[2*Y + 1] + tol) &&
            (rl[Z] >= bbox[2*Z + 0] - tol) && (rl[Z] < bbox[2*Z + 1] + tol);
    }
    
    static void bounce_1s(const Force *ff, const int np, const Mesh m, const float *bbox, /**/ Particle *pp, Solid *shst)
    {
        for (int i = 0; i < np; ++i)
        {
            Particle p = pp[i];
            if (in_bbox(p.r, shst->com, bbox, BBOX_MARGIN))
            {
                const Force f = ff[i];
                bounce_1s1p(f.f, m, /**/ &p, shst);
                pp[i] = p;
            }
        }
    }

    void bounce_hst(const Force *ff, const int np, const int ns, const Mesh m, const float *bboxes, /**/ Particle *pp, Solid *shst)
    {
#ifdef debug_output
        if (dstep % steps_per_dump == 0)
        for (int c = 0; c < 4; ++c) bbstates[c] = 0;
#endif

        for (int j = 0; j < ns; ++j)
        {
            Solid *s = shst + j;
            const float *bbox = bboxes + 6 * j;

            bounce_1s(ff, np, m, bbox, /**/ pp, s);
        }

#ifdef debug_output
        if ((++dstep) % steps_per_dump == 0)
        printf("%d success, %d nocross, %d wrong triangle, %d hfailed\n",
               bbstates[0], bbstates[1], bbstates[2], bbstates[3]);
#endif
    }
}
