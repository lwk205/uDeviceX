#include "hip/hip_runtime.h"
#include "sbounce.h"
#include <cassert>

namespace solidbounce {

    enum {X, Y, Z};
        
    // from forward Euler
    _DH_ void rvprev(const float *r1, const float *v1, const float *f0, /**/ float *r0, float *v0)
    {
        for (int c = 0; c < 3; ++c)
        {
            v0[c] = v1[c] - f0[c] * dt;
            r0[c] = r1[c] - v0[c] * dt;
        }
    }
    
    /*
      return true if a root h lies in [0, dt] (output h), false otherwise
      if 2 roots in [0, dt], smallest root h0 is returned (first collision)
    */
    _DH_ bool robust_quadratic_roots(const float a, const float b, const float c, /**/ float *h)
    {
        float D, h0, h1;
        int sgnb;

        sgnb = b > 0 ? 1 : -1;
        D = b*b - 4*a*c;

        if (D < 0) return false;
        
        h0 = (-b - sgnb * sqrt(D)) / (2 * a);
        h1 = c / (a * h0);
        
        if (h0 > h1)
        {
            float htmp = h1;
            h1 = h0; h0 = htmp;
        }

        if (h0 >= 0 && h0 <= dt) {*h = h0; return true;}
        if (h1 >= 0 && h1 <= dt) {*h = h1; return true;}
        
        return false;
    }

#if defined(rsph)

#define shape sphere

    namespace sphere
    {
        #define rsph_bb rsph

        _DH_ bool inside(const float *r)
        {
            return r[X] * r[X] + r[Y] * r[Y] + r[Z] * r[Z] < rsph_bb * rsph_bb;
        }

        _DH_ bool intersect(const float *r0, const float *v0, const float *vcm, const float *om0, /**/ float *h)
        {
            float r0x = r0[X],          r0y = r0[Y],          r0z = r0[Z];
            float v0x = v0[X] - vcm[X], v0y = v0[Y] - vcm[Y], v0z = v0[Z] - vcm[Z];
                        
            const float a = v0x*v0x + v0y*v0y + v0z*v0z;
            
            const float b = 2 * (r0x * v0x + r0y * v0y + r0z * v0z);
            const float c = r0x * r0x + r0y * r0y + r0z * r0z - rsph_bb * rsph_bb;
        
            return robust_quadratic_roots(a, b, c, h);
        }

        _DH_ void rescue(float *r)
        {
            float scale = (rsph_bb + 1e-6) / sqrt(r[X] * r[X] + r[Y] * r[Y] + r[Z] * r[Z]);

            r[X] *= scale;
            r[Y] *= scale;
            r[Z] *= scale;
        }
    }
    
#elif defined(rcyl)

#define shape cylinder
    
    namespace cylinder
    {   
        #define rcyl_bb rcyl

        _DH_ bool inside(const float *r)
        {
            return r[X] * r[X] + r[Y] * r[Y] < rcyl_bb * rcyl_bb;
        }

        /* output h between 0 and dt */
        _DH_ bool intersect(const float *r0, const float *v0, const float *vcm, const float *om0, /**/ float *h)
        {
            float r0x = r0[X],          r0y = r0[Y];
            float v0x = v0[X] - vcm[X], v0y = v0[Y] - vcm[Y];

            const float a = v0x * v0x + v0y * v0y;
            
            const float b = 2 * (r0x * v0x + r0y * v0y);
                        
            const float c = r0x * r0x + r0y * r0y - rcyl_bb * rcyl_bb;

            return robust_quadratic_roots(a, b, c, h);
        }

        _DH_ void rescue(float *r)
        {
            float scale = (rcyl_bb + 1e-6) / sqrt(r[X] * r[X] + r[Y] * r[Y]);

            r[X] *= scale;
            r[Y] *= scale;
        }
    }

#elif defined(a2_ellipse)

#define shape ellipse // "extruded" ellipse x^2/2^ + y^2/b^2 = 1
    
    namespace ellipse
    {
#define a2_bb a2_ellipse 
#define b2_bb b2_ellipse

        _DH_ bool inside(const float *r)
        {
            const float x = r[X];
            const float y = r[Y];
            
            return x*x / a2_bb + y*y / b2_bb < 1;
        }
        
        /* output h between 0 and dt */
        // for now: assume vcm = 0
        _DH_ bool intersect(const float *r0, const float *v0, const float *vcm, const float *om0, /**/ float *h)
        {
            const float r0x = r0[X],          r0y = r0[Y];
            const float v0x = v0[X] - vcm[X], v0y = v0[Y] - vcm[Y];

            const float om0z = -om0[Z];
            
            const float v0x_ = v0x - om0z * (r0y + dt * v0y);
            const float v0y_ = v0y + om0z * (r0x + dt * v0x);

            const float r0x_ = r0x + dt * om0z * (r0y + dt * v0y);
            const float r0y_ = r0y - dt * om0z * (r0x + dt * v0x);
            
            const float a = v0x_*v0x_ / a2_bb + v0y_*v0y_ / b2_bb;
            
            const float b = 2 * (r0x_ * v0x_ / a2_bb + r0y_ * v0y_ / b2_bb);
                        
            const float c = r0x_ * r0x_ / a2_bb + r0y_ * r0y_ / b2_bb - 1;

            return robust_quadratic_roots(a, b, c, h);
        }

        _DH_ void rescue(float *r) // cheap rescue
        {
            float scale = (1 + 1e-6) / sqrt(r[X] * r[X] / a2_bb + r[Y] * r[Y] / b2_bb);
            
            r[X] *= scale;
            r[Y] *= scale;
        }
    }

#elif defined(a2_ellipsoid)

#define shape ellipsoid
    
    namespace ellipsoid
    {
#define a2_bb a2_ellipsoid
#define b2_bb b2_ellipsoid
#define c2_bb c2_ellipsoid

        __DH__ bool inside(const float *r)
        {
            const float x = r[X];
            const float y = r[Y];
            const float z = r[Z];
            
            return x*x / a2_bb + y*y / b2_bb + z*z / c2_bb < 1;
        }

        _DH_ bool intersect(const float *r0, const float *v0, const float *vcm, const float *om0, /**/ float *h)
        {
            const float r0x  = r0[X],          r0   = r0[Y],          r0z  = r0[Z];
            const float v0x  = v0[X] - vcm[X], v0   = v0[Y] - vcm[Y], v0z  = v0[Z] - vcm[Z];
            const float om0x = om0[X],         om0y = om0[Y],         om0z = om0[Z];

            const float r1x = r0x + dt * v0x;
            const float r1y = r0y + dt * v0y;
            const float r1z = r0z + dt * v0z;
            
            const float v0x_ = v0x + om0y * r1z - om0z * r1y;
            const float v0y_ = v0y + om0z * r1x - om0x * r1z;
            const float v0z_ = v0z + om0x * r1y - om0y * r1x;

            const float r0x_ = r0x - dt * (om0z * r1z - om0z * r1y);
            const float r0y_ = r0y - dt * (om0z * r1x - om0x * r1z);
            const float r0z_ = r0z - dt * (om0x * r1y - om0y * r1x);
                
            
            const float a = v0x_*v0x_ / a2_bb + v0y_*v0y_ / b2_bb + v0z_*v0z_ / c2_bb;
            
            const float b = 2 * (r0x_*v0x_ / a2_bb + r0y_*v0y_ / b2_bb + r0z_*v0z_ / c2_bb);
                        
            const float c = r0x_*r0x_ / a2_bb + r0y_*r0y_ / b2_bb + r0z_*r0z_ / c2_bb - 1;

            return robust_quadratic_roots(a, b, c, h);
        }

        _DH_ void rescue(float *r) // cheap rescue
        {
            float scale = (1 + 1e-6) / sqrt(r[X] * r[X] / a2_bb + r[Y] * r[Y] / b2_bb + r[Z] * r[Z] / c2_bb);

            r[X] *= scale;
            r[Y] *= scale;
            r[Z] *= scale;
        }
    }
    
#else

#define shape none
    namespace none
    {
        _DH_ bool inside(const float *r)
        {
            printf("solidbounce: not implemented\n");
            exit(1);

            return false;
        }

        _DH_ bool intersect(const float *r0, const float *v0, const float *vcm, const float *om0, /**/ float *h)
        {
            printf("solidbounce: not implemented\n");
            exit(1);

            return false;
        }

        _DH_ void rescue(float *r)
        {
            printf("solidbounce: not implemented\n");
            exit(1);
        }
    }
    
#endif

    _DH_ bool inside_prev(const float *r, const float *om0)
    {
        float rl[3] = {float(r[X] + dt * (om0[Y] * r[Z] - om0[Z] * r[Y])),
                       float(r[Y] + dt * (om0[Z] * r[X] - om0[X] * r[Z])),
                       float(r[Z] + dt * (om0[X] * r[Y] - om0[Y] * r[X]))};
        
        return shape::inside(rl);
    }

    
    _DH_ void collision_point(const float *r0, const float *v0, const float h, /**/ float *rcol)
    {
        for (int c = 0; c < 3; ++c)
        rcol[c] = r0[c] + h * v0[c];
    }

    _DH_ void vsolid(const float *vcm, const float *om, const float *r, /**/ float *vs)
    {
        vs[X] = vcm[X] + om[Y]*r[Z] - om[Z]*r[Y];
        vs[Y] = vcm[Y] + om[Z]*r[X] - om[X]*r[Z];
        vs[Z] = vcm[Z] + om[X]*r[Y] - om[Y]*r[X];
    }

    _DH_ void bounce_particle(const float *vs, const float *rcol, const float *v0, const float h, /**/ float *rn, float *vn)
    {
        assert(h >= 0);
        assert(h <= dt);
        
        for (int c = 0; c < 3; ++c)
        {
            vn[c] = 2 * vs[c] - v0[c];
            rn[c] = rcol[c] + (dt - h) * vn[c];
        }
    }

    _DH_ void rescue_particle(const float *vcm, const float *om, /**/ float *r, float *v)
    {
        shape::rescue(/**/ r);
        vsolid(vcm, om, r, /**/ v);

        assert(!shape::inside(r));
    }

    _DH_ void lin_mom_solid(const float *v1, const float *vn, /**/ float *dP)
    {
        for (int c = 0; c < 3; ++c)
        dP[c] = -(vn[c] - v1[c]) / dt;
    }

    _DH_ void ang_mom_solid(const float *com, const float *r1, const float *rn, const float *v1, const float *vn, /**/ float *dL)
    {
        const float drn[3] = {rn[X] - com[X], rn[Y] - com[Y], rn[Z] - com[Z]};
        const float dr1[3] = {r1[X] - com[X], r1[Y] - com[Y], r1[Z] - com[Z]};
        
        dL[X] = -(drn[Y] * vn[Z] - drn[Z] * vn[Y] - dr1[Y] * v1[Z] + dr1[Z] * v1[Y]) / dt;
        dL[Y] = -(drn[Z] * vn[X] - drn[X] * vn[Z] - dr1[Z] * v1[X] + dr1[X] * v1[Z]) / dt;
        dL[Z] = -(drn[X] * vn[Y] - drn[Y] * vn[X] - dr1[X] * v1[Y] + dr1[Y] * v1[X]) / dt;
    }

#define debug_output
#ifdef debug_output
    int nrescued, nbounced, still_in, failed, step = 0;
    __device__ int bbinfosdev[5];
    FILE * fdebug;
#endif

    enum BBState
    {
        BB_SUCCESS,
        BB_RESCUED,
        BB_FAILED,
        BB_INSIDE,
        BB_NOBOUNCE
    };
    
    
    _DH_ BBState bb_part_local(const float *fp, const float *vcm, const float *om, /*o*/ Particle *p1, float *rw, float *vw, /*w*/ Particle *p0)
    {
        float h;
        
        if (!shape::inside(p1->r))
        return BB_NOBOUNCE;

        /* previous position and velocity                        */
        /* this step should be dependant on the time scheme only */
        
        rvprev(p1->r, p1->v, fp, /**/ p0->r, p0->v);

        /* rescue particles which were already in the solid   */
        /* put them back on the surface with surface velocity */

        if (inside_prev(p0->r, om))
        {
            rescue_particle(vcm, om, /**/ p1->r, p1->v);
            return BB_RESCUED;
        }
        
        /* find collision point */
        
        if (!shape::intersect(p0->r, p0->v, vcm, om, /**/ &h))
        return BB_FAILED;
        
        assert(h >= 0 );
        assert(h <= dt);
        
        collision_point(p0->r, p0->v, h, /**/ rw);
        
        /* handle collision for particle */
        
        vsolid(vcm, om, rw, /**/ vw);

        bounce_particle(vw, rw, p0->v, h, /**/ p1->r, p1->v);

        if (shape::inside(p1->r))
        return BB_INSIDE;

        return BB_SUCCESS;
    }
    
    _DH_ void r2local (const float *e0, const float *e1, const float *e2, const float *com, const float *rg, /**/ float *rl)
    {
        float x = rg[X] - com[X];
        float y = rg[Y] - com[Y];
        float z = rg[Z] - com[Z];
        
        rl[X] = x*e0[X] + y*e0[Y] + z*e0[Z];
        rl[Y] = x*e1[X] + y*e1[Y] + z*e1[Z];
        rl[Z] = x*e2[X] + y*e2[Y] + z*e2[Z];
    }

    _DH_ void r2global(const float *e0, const float *e1, const float *e2, const float *com, const float *rl, /**/ float *rg)
    {
        rg[X] = com[X] + rl[X] * e0[X] + rl[Y] * e1[X] + rl[Z] * e2[X];
        rg[Y] = com[Y] + rl[X] * e0[Y] + rl[Y] * e1[Y] + rl[Z] * e2[Y];
        rg[Z] = com[Z] + rl[X] * e0[Z] + rl[Y] * e1[Z] + rl[Z] * e2[Z];
    }

    _DH_ void v2local (const float *e0, const float *e1, const float *e2, const float *vg, /**/ float *vl)
    {
        vl[X] = vg[X]*e0[X] + vg[Y]*e0[Y] + vg[Z]*e0[Z];
        vl[Y] = vg[X]*e1[X] + vg[Y]*e1[Y] + vg[Z]*e1[Z];
        vl[Z] = vg[X]*e2[X] + vg[Y]*e2[Y] + vg[Z]*e2[Z];
    }

    _DH_ void v2global(const float *e0, const float *e1, const float *e2, const float *vl, /**/ float *vg)
    {
        vg[X] = vl[X] * e0[X] + vl[Y] * e1[X] + vl[Z] * e2[X];
        vg[Y] = vl[X] * e0[Y] + vl[Y] * e1[Y] + vl[Z] * e2[Y];
        vg[Z] = vl[X] * e0[Z] + vl[Y] * e1[Z] + vl[Z] * e2[Z];
    }
    
    void bounce(const Force *ff, const int np, /**/ Particle *pp, Solid *shst)
    {
        Particle p0l, p1, pn, pnl;
        float dP[3], dL[3], vcml[3], oml[3], fl[3], rw[3], vw[3];

#ifdef debug_output
        fdebug = fopen("debug.txt", "a");

        if (step % steps_per_dump == 0)
        nbounced = nrescued = still_in = failed = 0;
#endif
        
        for (int ip = 0; ip < np; ++ip)
        {
            p1 = pp[ip];
            pn = p1;

            r2local(shst->e0, shst->e1, shst->e2, shst->com, pn.r, /**/ pnl.r);
            v2local(shst->e0, shst->e1, shst->e2,            pn.v, /**/ pnl.v);
                
            v2local(shst->e0, shst->e1, shst->e2,  shst->v, /**/ vcml);
            v2local(shst->e0, shst->e1, shst->e2, shst->om, /**/  oml);
                
            v2local(shst->e0, shst->e1, shst->e2, ff[ip].f, /**/ fl);

#ifdef debug_output
            Particle p1l = pnl;
#endif
            
            BBState bbstate = bb_part_local(fl, vcml, oml, /*o*/ &pnl, rw, vw, /*w*/ &p0l);

#ifdef debug_output
            if (bbstate != BB_NOBOUNCE)
            {
#define db(...) fprintf (fdebug, __VA_ARGS__)
                db("%+.10e %+.10e %+.10e %+.10e %+.10e %+.10e ", p0l.r[X], p0l.r[Y], p0l.r[Z], p0l.v[X], p0l.v[Y], p0l.v[Z]);
                db("%+.10e %+.10e %+.10e %+.10e %+.10e %+.10e ", p1l.r[X], p1l.r[Y], p1l.r[Z], p1l.v[X], p1l.v[Y], p1l.v[Z]);
                db("%+.10e %+.10e %+.10e %+.10e %+.10e %+.10e ", rw[X], rw[Y], rw[Z], vw[X], vw[Y], vw[Z]);
                db("%+.10e %+.10e %+.10e %+.10e %+.10e %+.10e ", pnl.r[X], pnl.r[Y], pnl.r[Z], pnl.v[X], pnl.v[Y], pnl.v[Z]);

                switch (bbstate)
                {
                case BB_SUCCESS:
                    ++nbounced; db(":success:\n");
                    break;
                case BB_RESCUED:
                    ++nrescued; db(":rescued:\n");
                    break;
                case BB_INSIDE:
                    ++still_in; db(":inside:\n");
                    break;
                case BB_FAILED:
                    ++failed;   db(":failed:\n");
                    break;
                }
            }
#endif

            
            r2global(shst->e0, shst->e1, shst->e2, shst->com, pnl.r, /**/ pn.r);
            v2global(shst->e0, shst->e1, shst->e2,            pnl.v, /**/ pn.v); 
            
            /* transfer momentum */
            
            dP[X] = dP[Y] = dP[Z] = 0;
            dL[X] = dL[Y] = dL[Z] = 0;
                
            lin_mom_solid(p1.v, pn.v, /**/ dP);
                
            ang_mom_solid(shst->com, p1.r, pn.r, p1.v, pn.v, /**/ dL);
                
            for (int d = 0; d < 3; ++d)
            {
                shst->fo[d] += dP[d];
                shst->to[d] += dL[d];
            }

            pp[ip] = pn;
        }
#ifdef debug_output
        if ((++step) % steps_per_dump == 0)
        printf("%d rescued, %d boounced, %d still in, %d failed\n\n", nrescued, nbounced, still_in, failed);

        fclose(fdebug);
#endif
    }

    __device__ void warpReduceSumf3(float *x)
    {
        for (int offset = warpSize>>1; offset > 0; offset >>= 1)
        {
            x[X] += __shfl_down(x[X], offset);
            x[Y] += __shfl_down(x[Y], offset);
            x[Z] += __shfl_down(x[Z], offset);
        }
    }

    __global__ void bounce_kernel(const Force *ff, const int np, /**/ Particle *pp, Solid *sdev)
    {
        const int pid = threadIdx.x + blockDim.x * blockIdx.x;

        float dP[3] = {0.f, 0.f, 0.f};
        float dL[3] = {0.f, 0.f, 0.f};

        if (pid < np)
        {
            Particle p0l, p1, pn, pnl;
            float vcml[3], oml[3], fl[3], rw[3], vw[3];
            
            p1 = pp[pid];
            pn = p1;

            r2local(sdev->e0, sdev->e1, sdev->e2, sdev->com, pn.r, /**/ pnl.r);
            v2local(sdev->e0, sdev->e1, sdev->e2,            pn.v, /**/ pnl.v);
                
            v2local(sdev->e0, sdev->e1, sdev->e2,  sdev->v, /**/ vcml);
            v2local(sdev->e0, sdev->e1, sdev->e2, sdev->om, /**/  oml);
                
            v2local(sdev->e0, sdev->e1, sdev->e2, ff[pid].f, /**/ fl);
                
            BBState bbstate = bb_part_local(fl, vcml, oml, /*o*/ &pnl, rw, vw, /*w*/ &p0l);

#ifdef debug_output
            if (bbstate != BB_NOBOUNCE) atomicAdd(bbinfosdev + bbstate, 1);
#endif
            
            r2global(sdev->e0, sdev->e1, sdev->e2, sdev->com, pnl.r, /**/ pn.r);
            v2global(sdev->e0, sdev->e1, sdev->e2,            pnl.v, /**/ pn.v); 
                
            /* transfer momentum */

            lin_mom_solid(p1.v, pn.v, /**/ dP);
                
            ang_mom_solid(sdev->com, p1.r, pn.r, p1.v, pn.v, /**/ dL);
            
            pp[pid] = pn;
        }

        /* momentum reduction */
        
        warpReduceSumf3(dP);
        warpReduceSumf3(dL);

        const float normdP = fmaxf(fmaxf(fabsf(dP[X]), fabsf(dP[Y])), fabsf(dP[Z]));
        const float normdL = fmaxf(fmaxf(fabsf(dL[X]), fabsf(dL[Y])), fabsf(dL[Z]));

        const bool warp_contribute = (normdP > 1e-12) && (normdL > 1e-12);
        
        if (warp_contribute && ((threadIdx.x & (warpSize - 1)) == 0))
        {
            atomicAdd(sdev->fo + X, dP[X]);
            atomicAdd(sdev->fo + Y, dP[Y]);
            atomicAdd(sdev->fo + Z, dP[Z]);

            atomicAdd(sdev->to + X, dL[X]);
            atomicAdd(sdev->to + Y, dL[Y]);
            atomicAdd(sdev->to + Z, dL[Z]);
        }
    }

    void bounce_nohost(const Force *ff, const int np, /**/ Particle *pp, Solid *sdev)
    {
#ifdef debug_output
        if (step % steps_per_dump == 0)
        {
            const int zeros[5] = {0, 0, 0, 0, 0};
            CC(hipMemcpyToSymbol(HIP_SYMBOL(bbinfosdev), zeros, 5*sizeof(int)));
        }
#endif

        bounce_kernel <<<k_cnf(np)>>> (ff, np, /**/ pp, sdev);

#ifdef debug_output
        if ((++step) % steps_per_dump == 0)
        {
            int bbinfos[5];
            CC(hipMemcpyFromSymbol(bbinfos, HIP_SYMBOL(bbinfosdev), 5*sizeof(int)));
            
            printf("%d rescued, %d boounced, %d still in, %d failed\n\n", bbinfos[BB_RESCUED], bbinfos[BB_SUCCESS], bbinfos[BB_INSIDE], bbinfos[BB_FAILED]);
        }
#endif

    }
}
