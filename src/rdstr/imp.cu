#include "hip/hip_runtime.h"
#include <stdio.h>
#include <mpi.h>

#include <conf.h>
#include "inc/conf.h"

#include "d/q.h"
#include "d/api.h"
#include "d/ker.h"

#include "inc/def.h"
#include "msg.h"
#include "m.h"
#include "cc.h"
#include "l/m.h"

#include "inc/type.h"
#include "inc/mpi.h"
#include "inc/dev.h"
#include "mc.h"

#include "kl.h"

#include "minmax.h"

#include "mdstr/buf.h"
#include "mdstr/gen.h"

#include "rdstr/imp.h"
#include "rdstr/dev.h"

namespace rdstr {
namespace sub {

enum {X, Y, Z};

void waitall(MPI_Request rr[26]) {
    MPI_Status ss[26];
    l::m::Waitall(26, rr, ss) ;
}

void cancelall(MPI_Request rr[26]) {
    for (int i = 0; i < 26; ++i) MC(MPI_Cancel(rr + i));
}

void extents(const Particle *pp, int nc, int nv, /**/ float3 *ll, float3 *hh) {
    if (nc) minmax(pp, nv, nc, /**/ ll, hh);
}

void get_pos(int n, const float3 *ll, const float3 *hh, /**/ float *rr) {
    for (int i = 0; i < n; ++i) {
        float3 l = ll[i], h = hh[i];
        float *r = rr + 3 * i;
        r[X] = 0.5f * (l.x + h.x);
        r[Y] = 0.5f * (l.y + h.y);
        r[Z] = 0.5f * (l.z + h.z);
    }
}

void pack(int *reord[27], const int counts[27], const Particle *pp, int nv, /**/ Partbuf *bpp) {
    gen::pack <Particle, gen::Device> (reord, counts, pp, nv, /**/ bpp);
}

void post_send(int nv, const int counts[27], const Partbuf *bpp, MPI_Comm cart, int bt, int rnk_ne[27],
               /**/ MPI_Request sreq[26]) {
    dSync(); // wait for pack
    gen::post_send(nv, counts, bpp, cart, bt, rnk_ne, /**/ sreq);
}

void post_recv(MPI_Comm cart, int nmax, int bt, int ank_ne[27], /**/ Partbuf *bpp, MPI_Request rreq[26]) {
    gen::post_recv(cart, nmax, bt, ank_ne, /**/ bpp, rreq);
}

int unpack(int npd, const Partbuf *bpp, const int counts[27], /**/ Particle *pp) {
    return gen::unpack <Particle, gen::Device> (npd, bpp, counts, /**/ pp);
}

void shift(int npd, const int counts[27], /**/ Particle *pp) {
    int nm = counts[0]; /* skip bulk */
    for (int i = 1; i < 27; ++i) {
        int c = counts[i];
        int n = c * npd;
        KL(dev::shift, (k_cnf(n)), (n, i, /**/ pp + nm * npd));
        nm += c;
    }
}

} // sub
} // rdstr
