#include "hip/hip_runtime.h"
#include "common.h"
#include "minmax.h"

#include "mesh.h"

namespace mesh
{
    enum {X, Y, Z};
    #define _HD_ __host__ __device__

    static _HD_ bool same_side(const float *x, const float *p, const float *a, const float *b, const float *inplane)
    {
        const float n[3] = {a[Y] * b[Z] - a[Z] * b[Y],
                            a[Z] * b[X] - a[X] * b[Z],
                            a[X] * b[Y] - a[Y] * b[X]};

        const float ndx = n[X] * (x[X] - inplane[X]) + n[Y] * (x[Y] - inplane[Y]) + n[Z] * (x[Z] - inplane[Z]);
        const float ndp = n[X] * (p[X] - inplane[X]) + n[Y] * (p[Y] - inplane[Y]) + n[Z] * (p[Z] - inplane[Z]);
        
        return ndx * ndp > 0;
    }
    
    static _HD_ bool in_tetrahedron(const float *x, const float *A, const float *B, const float *C, const float *D)
    {
        const float AB[3] = {B[X] - A[X], B[Y] - A[Y], B[Z] - A[Z]};
        const float AC[3] = {C[X] - A[X], C[Y] - A[Y], C[Z] - A[Z]};
        const float AD[3] = {D[X] - A[X], D[Y] - A[Y], D[Z] - A[Z]};
    
        const float BC[3] = {C[X] - B[X], C[Y] - B[Y], C[Z] - B[Z]};
        const float BD[3] = {D[X] - B[X], D[Y] - B[Y], D[Z] - B[Z]};

        return
            same_side(x, A, BC, BD, D) &&
            same_side(x, B, AD, AC, D) &&
            same_side(x, C, AB, BD, D) &&
            same_side(x, D, AB, AC, A);
    }
    
    int inside_1p(const float *r, const float *vv, const int *tt, const int nt)
    {
        int c = 0;
        const float origin[3] = {0, 0, 0};
        for (int i = 0; i < nt; ++i)
        {
            const int *t = tt + 3 * i;
            if (in_tetrahedron(r, vv + 3*t[0], vv + 3*t[1], vv + 3*t[2], origin)) ++c;
        }
        return c%2;
    }

    void inside_hst(const Particle *pp, const int n, const Mesh m, /**/ int *inout)
    {
        for (int i = 0; i < n; ++i)
        inout[i] = inside_1p(pp[i].r, m.vv, m.tt, m.nt);
    }

    static int inside_1p(const float *r, const Particle *vv, const int *tt, const int nt)
    {
        int c = 0;
        const float origin[3] = {0, 0, 0};
        for (int i = 0; i < nt; ++i)
        {
            const int *t = tt + 3 * i;
            if (in_tetrahedron(r, vv[t[0]].r, vv[t[1]].r, vv[t[2]].r, origin)) ++c;
        }
        return c%2;
    }
    
    void inside_hst(const Particle *pp, const int n, const Mesh m, const Particle *i_pp, const int ns, /**/ int *tags)
    {
        for (int i = 0; i < n; ++i)
        {
            tags[i] = -1;
            for (int sid = 0; sid < ns; ++sid)
            if (inside_1p(pp[i].r, i_pp + m.nv * sid, m.tt, m.nt))
            {
                tags[i] = sid;
                break;
            }
        }
    }

    namespace kernels
    {
        __global__ void init_tags(const int n, /**/ int *tags)
        {
            const int gid = threadIdx.x + blockIdx.x * blockDim.x;
            if (gid < n) tags[gid] = -1;
        }

        // assume ns blocks along y
        __global__ void compute_tags(const Particle *pp, const int n, const Particle *vv, const int nv, const int *tt, const int nt, /**/ int *tags)
        {
            const int sid = blockIdx.y;
            const int gid = threadIdx.x + blockIdx.x * blockDim.x;
            if (gid >= n) return;

            int count = 0;

            const Particle p = pp[gid];
            const float origin[3] = {0, 0, 0};
        
            for (int i = 0; i < nt; ++i)
            {
                const int t1 = sid * nv + tt[3*i + 0];
                const int t2 = sid * nv + tt[3*i + 1];
                const int t3 = sid * nv + tt[3*i + 2];

                const float a[3] = {vv[t1].r[0], vv[t1].r[1], vv[t1].r[2]};
                const float b[3] = {vv[t2].r[0], vv[t2].r[1], vv[t2].r[2]};
                const float c[3] = {vv[t3].r[0], vv[t3].r[1], vv[t3].r[2]};
            
                if (in_tetrahedron(p.r, a, b, c, origin)) ++count;
            }

            // dont consider the case of inside several solids
            if (count % 2) atomicExch(tags + gid, sid);
        }
    }
    
    void inside_dev(const Particle *pp, const int n, const Mesh m, const Particle *i_pp, const int ns, /**/ int *tags)
    {
        if (ns == 0 || n == 0) return;
        
        kernels::init_tags <<< k_cnf(n) >>> (n, /**/ tags);

        dim3 thrd(128, 1);
        dim3 blck((127 + n)/128, ns);

        kernels::compute_tags <<< blck, thrd >>> (pp, n, i_pp, m.nv, m.tt, m.nt, /**/ tags);
    }

    /* bbox: minx, maxx, miny, maxy, minz, maxz */
    static void get_bbox(const Particle *pp, const int n, /**/ float *bbox)
    {
        if (n == 0) return;

        const float *r = pp[0].r;
        
        bbox[0] = bbox[1] = r[0];
        bbox[2] = bbox[3] = r[1];
        bbox[4] = bbox[5] = r[2];

        auto higher = [](float a, float b) {return a > b ? a : b;};
        auto lower  = [](float a, float b) {return a > b ? b : a;};

        for (int i = 1; i < n; ++i)
        {
            r = pp[i].r;
            for (int d = 0; d < 3; ++d)
            {
                bbox[2*d + 0] =  lower(bbox[2*d + 0], r[d]);
                bbox[2*d + 1] = higher(bbox[2*d + 1], r[d]); 
            }
        }
    }

    void get_bbox(const float *rr, const int n, /**/ float *bbox)
    {
        if (n == 0) return;

        const float *r = rr;
        
        bbox[0] = bbox[1] = r[0];
        bbox[2] = bbox[3] = r[1];
        bbox[4] = bbox[5] = r[2];

        auto higher = [](float a, float b) {return a > b ? a : b;};
        auto lower  = [](float a, float b) {return a > b ? b : a;};

        for (int i = 1; i < n; ++i)
        {
            r = rr + 3 * i;;
            for (int d = 0; d < 3; ++d)
            {
                bbox[2*d + 0] =  lower(bbox[2*d + 0], r[d]);
                bbox[2*d + 1] = higher(bbox[2*d + 1], r[d]); 
            }
        }
    }

    void get_bboxes_hst(const Particle *pp, const int nps, const int ns, /**/ float *bboxes)
    {
        for (int i = 0; i < ns; ++i)
        get_bbox(pp + i*nps, nps, /**/ bboxes + 6 * i);
    }

    void get_bboxes_dev(const Particle *pp, const int nps, const int ns, /**/ float *bboxes)
    {
        if (ns == 0) return;
        minmax(pp, nps, ns, /**/ bboxes);
    }
}
