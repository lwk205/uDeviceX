#include <stdio.h>
#include <conf.h>

#include "inc/conf.h"
#include "msg.h"
#include "cc.h"
#include "d/api.h"
#include "inc/dev.h"

#include "dual/type.h"
#include "dual/int.h"

namespace dual {
void alloc(I *p, int n) {
    Palloc0(&p->D, n);
    Link(&p->DP, p->D);
}

void dealloc(I p) {
    int *D;
    D = p.D;
    CC(hipHostFree(D));
}
}
