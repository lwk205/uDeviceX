#include <stdio.h>
#include <conf.h>

#include "inc/conf.h"
#include "msg.h"
#include "cc.h"
#include "d.h"
#include "inc/dev.h"

#include "dual/type.h"
#include "dual/int.h"

namespace dual {
void alloc(I *p, int n) {
    Palloc0(&p->D, n);
    CC(hipHostGetDevicePointer(&p->DP, p->D, 0));
}

void dealloc(I p) {
    int *D;
    D = p.D;
    CC(hipHostFree(D));
}
}
