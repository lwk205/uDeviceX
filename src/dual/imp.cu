#include <stdio.h>
#include <conf.h>

#include "inc/conf.h"
#include "msg.h"
#include "cc.h"
#include "inc/dev.h"

#include "dual/type.h"
#include "dual/int.h"

namespace dual {
void alloc(I *p, int n) {
    CC(hipHostAlloc(&p->D, sizeof(int) * n, hipHostMallocMapped));
    CC(hipHostGetDevicePointer(&p->DP, p->D, 0));
}

void dealloc(I p) {
    int *D;
    D = p.D;
    CC(hipHostFree(D));
}
}
