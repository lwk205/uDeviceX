#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"

#include "inc/def.h"
#include "msg.h"

#include "m.h"
#include "cc.h"
#include "d/api.h"
#include "d/q.h"
#include "d/ker.h"
#include "inc/type.h"
#include "inc/dev.h"

#include "kl.h"

// #define debug_output

#include "mbounce/imp.h"
#include "mbounce/bbstates.h"
#include "mbounce/dbg.h"
#include "mbounce/roots.h"
#include "mbounce/gen.h"
#include "mbounce/gen.intersect.h"
#include "mbounce/gen.tri.h"
#include "mbounce/hst.h"
#include "mbounce/dev.h"

namespace mbounce {

void alloc_ticketM(TicketM *t) {
    CC(hipMalloc(&t->mm_dev, MAX_PART_NUM * sizeof(Momentum)));
    t->mm_hst = new Momentum[MAX_PART_NUM];
}

void free_ticketM(TicketM *t) {
    CC(hipFree(t->mm_dev));
    delete[] t->mm_hst;
}

void bounce_hst(const Force *ff, const Mesh m, const Particle *i_pp, const int *tcellstarts, const int *tcellcounts, const int *tids,
                const int n, const int totnt, /**/ Particle *pp, TicketM *t) {
    sub::dbg::ini_hst();
    
    if (totnt && n) {
        memset(t->mm_hst, 0, totnt * sizeof(Momentum));
        sub::hst::bounce(ff, m.tt, m.nt, m.nv, i_pp, tcellstarts, tcellcounts, tids, n, /**/ pp, t->mm_hst);
    }
    
    sub::dbg::report_hst();
}

void bounce_dev(const Force *ff, const Mesh m, const Particle *i_pp, const int *tcellstarts, const int *tcellcounts, const int *tids,
                const int n, const int totnt, /**/ Particle *pp, TicketM *t) {
    sub::dbg::ini_dev();
    
    if (totnt && n) {
        CC(hipMemsetAsync(t->mm_dev, 0, totnt * sizeof(Momentum)));        
        KL(sub::dev::bounce,
           (k_cnf(n)),
           (ff, m.tt, m.nt, m.nv, i_pp, tcellstarts, tcellcounts, tids, n, /**/ pp, t->mm_dev));
    }
    
    sub::dbg::report_dev();
}

void bounce_rbc_hst(const Force *ff, const int4 *tt, int nt, int nv, const Particle *i_pp, const int *tcellstarts, const int *tcellcounts,
                    const int *tids, const int n, const int totnt, /**/ Particle *pp, TicketM *t) {
    sub::dbg::ini_hst();
    
    if (totnt && n) {
        memset(t->mm_hst, 0, totnt * sizeof(Momentum));
        sub::hst::bounce(ff, tt, nt, nv, i_pp, tcellstarts, tcellcounts, tids, n, /**/ pp, t->mm_hst);
    }
    
    sub::dbg::report_hst();
}

void bounce_rbc_dev(const Force *ff, const int4 *tt, int nt, int nv, const Particle *i_pp, const int *tcellstarts, const int *tcellcounts,
                    const int *tids, const int n, const int totnt, /**/ Particle *pp, TicketM *t) {
    sub::dbg::ini_dev();
    
    if (totnt && n) {
        CC(hipMemsetAsync(t->mm_dev, 0, totnt * sizeof(Momentum)));        
        KL(sub::dev::bounce,
           (k_cnf(n)),
           (ff, tt, nt, nv, i_pp, tcellstarts, tcellcounts, tids, n, /**/ pp, t->mm_dev));
    }
    
    sub::dbg::report_dev();
}

void collect_rig_hst(int nt, int ns, const TicketM *t, /**/ Solid *ss) {
    int n = ns * nt;
    if (n) sub::hst::collect_rig_mom (t->mm_hst, ns, nt, /**/ ss);
}

void collect_rig_dev(int nt, int ns, const TicketM *t, /**/ Solid *ss) {
    int n = ns * nt;
    KL(sub::dev::collect_rig_mom,
       (k_cnf(n)),
       (t->mm_dev, ns, nt, /**/ ss));
}

} // mbounce

#ifdef debug_output
#undef debug_output
#endif
