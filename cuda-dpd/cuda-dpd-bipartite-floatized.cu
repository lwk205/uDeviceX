#include "hip/hip_runtime.h"
#include "dpd-rng.h"
#include "hacks.h"
#include "../src/dpd-forces.h"

struct BipartiteInfoDPD {
    int3 ncells;
    float3 domainsize, invdomainsize, domainstart;
    float invrc;
};

__constant__ BipartiteInfoDPD bipart_info;

const static uint COLS = 8;
const static uint ROWS = (32 / COLS);
const static uint CPB = 4;

__global__
void _bipartite_dpd_directforces_floatized( float * const axayaz, const int np, const int np_src,
                                  const float seed, const int mask, const float * xyzuvw, const float * xyzuvw_src,
                                  const float invrc)
{
    const int tid = threadIdx.x % warpSize;
    const int pid = threadIdx.x + blockDim.x * blockIdx.x;
    const bool valid = pid < np;

    float xp, yp, zp, up, vp, wp;

    if( valid ) {
        xp = xyzuvw[0 + pid * 6];
        yp = xyzuvw[1 + pid * 6];
        zp = xyzuvw[2 + pid * 6];
        up = xyzuvw[3 + pid * 6];
        vp = xyzuvw[4 + pid * 6];
        wp = xyzuvw[5 + pid * 6];
    }

    float xforce = 0, yforce = 0, zforce = 0;

    for( int s = 0; s < np_src; s += warpSize ) {
        float my_xq, my_yq, my_zq, my_uq, my_vq, my_wq;

        const int batchsize = min( warpSize, np_src - s );

        if( tid < batchsize ) {
            my_xq = xyzuvw_src[0 + ( tid + s ) * 6];
            my_yq = xyzuvw_src[1 + ( tid + s ) * 6];
            my_zq = xyzuvw_src[2 + ( tid + s ) * 6];
            my_uq = xyzuvw_src[3 + ( tid + s ) * 6];
            my_vq = xyzuvw_src[4 + ( tid + s ) * 6];
            my_wq = xyzuvw_src[5 + ( tid + s ) * 6];
        }

        for( int l = 0; l < batchsize; ++l ) {
            const float xq = __shfl( my_xq, l );
            const float yq = __shfl( my_yq, l );
            const float zq = __shfl( my_zq, l );
            const float uq = __shfl( my_uq, l );
            const float vq = __shfl( my_vq, l );
            const float wq = __shfl( my_wq, l );

            {
                const int spid = s + l;
                const int dpid = pid;
                const int arg1 = mask * dpid + ( 1 - mask ) * spid;
                const int arg2 = mask * spid + ( 1 - mask ) * dpid;
                const float myrandnr = Logistic::mean0var1( seed, arg1, arg2 );

                // check for particle types and compute the DPD force
                float3 pos1 = make_float3(xp, yp, zp), pos2 = make_float3(xq, yq, zq);
                float3 vel1 = make_float3(up, vp, wp), vel2 = make_float3(uq, vq, wq);

                const float3 strength = compute_dpd_force_traced(SOLVENT_TYPE, SOLVENT_TYPE,
                        pos1, pos2, vel1, vel2, myrandnr);

                {
                    xforce += strength.x;
                    yforce += strength.y;
                    zforce += strength.z;
                }
            }
        }
    }

    if( valid ) {
        axayaz[0 + 3 * pid] = xforce;
        axayaz[1 + 3 * pid] = yforce;
        axayaz[2 + 3 * pid] = zforce;
    }
}

void directforces_dpd_cuda_bipartite_nohost(
    const float * const xyzuvw, float * const axayaz, const int np,
    const float * const xyzuvw_src, const int np_src,
    const float invsqrtdt,
    const float seed, const int mask)
{
    if( np == 0 || np_src == 0 ) {
        printf( "warning: directforces_dpd_cuda_bipartite_nohost called with ZERO!\n" );
        return;
    }

    _bipartite_dpd_directforces_floatized <<< ( np + 127 ) / 128, 128, 0 >>> ( axayaz, np, np_src, seed, mask,
            xyzuvw, xyzuvw_src, 1);

    CC( hipPeekAtLastError() );
}

__global__
void _dpd_bipforces_floatized( const float2 * const xyzuvw, const int np, hipTextureObject_t texDstStart,
                     hipTextureObject_t texSrcStart,  hipTextureObject_t texSrcParticles, const int np_src, const int3 halo_ncells,
                     const float seed, const uint mask, float * const axayaz )
{
    const uint tid = threadIdx.x;
    const uint subtid = tid % COLS;
    const uint slot = tid / COLS;
    const uint wid = threadIdx.y;

    __shared__ uint volatile starts[CPB][32], scan[CPB][32];

    const int mycid = xmad( blockIdx.x, float(CPB), threadIdx.y );

    if( mycid >= halo_ncells.x * halo_ncells.y * halo_ncells.z ) return;

    uint mycount = 0, myscan = 0;
    if( tid < 27 ) {
        const int dx = ( tid ) % 3;
        const int dy = ( ( tid / 3 ) ) % 3;
        const int dz = ( ( tid / 9 ) ) % 3;

        int xcid = ( mycid % halo_ncells.x ) + dx - 1;
        int ycid = ( ( mycid / halo_ncells.x ) % halo_ncells.y ) + dy - 1;
        int zcid = ( ( mycid / halo_ncells.x / halo_ncells.y ) % halo_ncells.z ) + dz - 1;

        const bool valid_cid =
                ( xcid >= 0 ) && ( xcid < halo_ncells.x ) &&
                ( ycid >= 0 ) && ( ycid < halo_ncells.y ) &&
                ( zcid >= 0 ) && ( zcid < halo_ncells.z );

        xcid = xmin( xsub( halo_ncells.x, 1 ), max( 0, xcid ) );
        ycid = xmin( xsub( halo_ncells.y, 1 ), max( 0, ycid ) );
        zcid = xmin( xsub( halo_ncells.z, 1 ), max( 0, zcid ) );

        const int cid = max( 0, xcid + halo_ncells.x * ( ycid + halo_ncells.y * zcid ) );

        starts[wid][tid] = tex1Dfetch<uint>( texSrcStart, cid );

        myscan = mycount = valid_cid ? ( tex1Dfetch<uint>( texSrcStart, cid + 1 ) - tex1Dfetch<uint>( texSrcStart, cid ) ) : 0u;
    }

    for( int L = 1; L < 32; L <<= 1 )
	//int or float yuhang?
        myscan += ( tid >= L ) * __shfl_up((int) myscan, L ) ;

    if( tid < 28 )
        scan[wid][tid] = myscan - mycount;

    const uint dststart = tex1Dfetch<uint>( texDstStart, mycid );
    const uint nsrc = scan[wid][27];
    const uint ndst = xsub( tex1Dfetch<uint>( texDstStart, xadd( mycid, 1 ) ), tex1Dfetch<uint>( texDstStart, mycid ) );

    for( uint d = 0u; d < ndst; d += ROWS ) {
        const uint np1 = xmin( ndst - d, ROWS );
        const uint dpid = xadd( xadd( dststart, d ), slot );
        const int entry = xscale( dpid, 3.f );
        float2 dtmp0 = xyzuvw[entry];
        float2 dtmp1 = xyzuvw[xadd(entry, 1)];
        float2 dtmp2 = xyzuvw[xadd(entry, 2)];

        float3 f = make_float3(0,0,0);

        for( uint s = 0; s < nsrc; s += COLS ) {
            const uint np2 = xmin( xsub(nsrc, s), COLS );
            const uint pid = xadd( s, subtid );
            const uint key9 = xadd( xsel_ge( pid, scan[wid][9            ], 9u, 0u ), xsel_ge( pid, scan[wid][18           ], 9u, 0u ) );
            const uint key3 = xadd( xsel_ge( pid, scan[wid][xadd(key9,3u)], 3u, 0u ), xsel_ge( pid, scan[wid][xadd(key9,6u)], 3u, 0u ) );
            const uint key  = xadd( key9, key3 );
            const uint spid = xsub( xadd( starts[wid][key], pid ), scan[wid][key] );
            const int sentry = xscale( spid, 3.f );

            const float2 stmp0 = tex1Dfetch<float2>( texSrcParticles, sentry );
            const float2 stmp1 = tex1Dfetch<float2>( texSrcParticles, xadd( sentry, 1 ) );
            const float2 stmp2 = tex1Dfetch<float2>( texSrcParticles, xadd( sentry, 2 ) );

            {
                const uint arg1 = xsel_gt( mask, 0u, dpid, spid );
                const uint arg2 = xsel_gt( mask, 0u, spid, dpid );
                const float myrandnr = Logistic::mean0var1( seed, arg1, arg2 );

                // check for particle types and compute the DPD force
                float3 pos1 = make_float3(dtmp0.x, dtmp0.y, dtmp1.x), pos2 = make_float3(stmp0.x, stmp0.y, stmp1.x);
                float3 vel1 = make_float3(dtmp1.y, dtmp2.x, dtmp2.y), vel2 = make_float3(stmp1.y, stmp2.x, stmp2.y);

                const float3 strength = compute_dpd_force_traced(SOLVENT_TYPE, SOLVENT_TYPE,
                        pos1, pos2, vel1, vel2, myrandnr);

                const bool valid = xfcmp_lt( slot, np1 ) * xfcmp_lt( subtid, np2 );

                if( valid ) {
                    f.x += strength.x;
                    f.y += strength.y;
                    f.z += strength.z;
                }
            }
        }

        for( uint L = COLS / 2u; L > 0u; L >>= 1 ) {
			f.x += __shfl_xor( f.x, L );
			f.y += __shfl_xor( f.y, L );
			f.z += __shfl_xor( f.z, L );
        }

        const uint c = ( subtid % 3u );
        const float fcontrib = xsel_eq( c, 0u, f.x, xsel_eq( c, 1u, f.y, f.z ) );
        const uint dstpid = xadd( xadd( dststart, d ), slot );

        if( slot < np1 )
            axayaz[ xmad( dstpid, 3.f, c ) ] = fcontrib;
    }
}

void forces_dpd_cuda_bipartite_nohost( const float2 * const xyzuvw, const int np, hipTextureObject_t texDstStart,
                                       hipTextureObject_t texSrcStart, hipTextureObject_t texSrcParticles, const int np_src,
                                       const int3 halo_ncells,
                                       const float seed, const int mask, float * const axayaz )
{
    const int ncells = halo_ncells.x * halo_ncells.y * halo_ncells.z;

    static bool fbip_init = false;

    if( !fbip_init ) {
        CC( hipFuncSetCacheConfig(reinterpret_cast<const void*>( _dpd_bipforces_floatized), hipFuncCachePreferL1 ) );

        fbip_init = true;
    }

    _dpd_bipforces_floatized <<< ( ncells + CPB - 1 ) / CPB, dim3( 32, CPB ), 0>>> (
        xyzuvw, np, texDstStart, texSrcStart, texSrcParticles, np_src,
        halo_ncells, seed, mask,
        axayaz );
}
